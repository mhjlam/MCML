#include "hip/hip_runtime.h"
/*==============================================================================
 * CUDAMCML Transport Module - GPU-Accelerated Photon Transport in Layered Media
 *
 * This module implements the core Monte Carlo photon transport physics for
 * multi-layered biological tissues. Provides massively parallel GPU kernels
 * for simulating photon propagation, scattering, absorption, and boundary
 * interactions with high computational efficiency.
 *
 * PHYSICAL MODEL IMPLEMENTATION:
 * ------------------------------
 * - Monte Carlo photon transport with statistical weight tracking
 * - Multi-layered geometry with arbitrary optical properties per layer
 * - Henyey-Greenberg scattering phase function for anisotropic scattering
 * - Fresnel reflection/transmission at all refractive index boundaries
 * - Beer-Lambert absorption with layer-specific absorption coefficients
 * - Russian roulette photon termination for computational efficiency
 *
 * GPU COMPUTATIONAL ARCHITECTURE:
 * -------------------------------
 * - One photon per GPU thread for massive parallelization
 * - Coalesced memory access patterns for optimal bandwidth utilization
 * - Template specialization for performance optimization (absorption detection)
 * - Atomic operations for thread-safe detection accumulation
 * - Efficient random number generation with per-thread state management
 *
 * DETECTION CAPABILITIES:
 * -----------------------
 * - Reflectance: Rd(r,α) - spatially and angularly resolved top surface detection
 * - Absorption: A(r,z) - volumetric energy deposition throughout tissue layers
 * - Transmittance: Tt(r,α) - spatially and angularly resolved bottom surface detection
 * - Real-time photon counting and statistical accumulation
 *
 * PERFORMANCE OPTIMIZATIONS:
 * ---------------------------
 * - Template specialization for compile-time optimization paths
 * - Single-precision floating-point arithmetic for GPU efficiency
 * - Minimized divergent branching in hot computation paths
 * - Efficient use of GPU constant memory for frequently accessed parameters
 * - Optimized atomic operations for detection data accumulation
 *
 * LICENSE:
 * --------
 * This file is part of CUDAMCML.
 *
 * CUDAMCML is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * CUDAMCML is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with CUDAMCML.  If not, see <http://www.gnu.org/licenses/>.
 */

////////////////////////////////////////////////////////////////////////////////
// INCLUDES AND DEPENDENCIES

// Standard library includes
#include <cstdint> // Standard integer types for cross-platform compatibility

////////////////////////////////////////////////////////////////////////////////
// DEVICE FUNCTION FORWARD DECLARATIONS

/**
 * Forward declarations of device functions for proper compilation ordering.
 * These functions implement the core physics of photon transport and are
 * called extensively within the main transport kernels.
 */

// Core transport kernel (template for performance optimization)
template<int ignoreAdetection>
__global__ void mc_d(MemStruct DeviceMem);

// Random number generation functions (imported from RNG module)
__device__ float rand_mwc_oc(uint64_t* rng_state, uint32_t* rng_multiplier); // (0,1] interval
__device__ float rand_mwc_co(uint64_t* rng_state, uint32_t* rng_multiplier); // [0,1) interval

// Photon lifecycle management
__device__ void launch_photon(PhotonStruct* photon_state, uint64_t* rng_state, uint32_t* rng_multiplier);
__global__ void launch_photon_global(MemStruct device_memory);
__device__ uint32_t photon_survive(PhotonStruct* photon_state, uint64_t* rng_state, uint32_t* rng_multiplier);

// Physics implementations
__device__ void spin(PhotonStruct* photon_state, float anisotropy_g, uint64_t* rng_state, uint32_t* rng_multiplier);
__device__ uint32_t reflect(PhotonStruct* photon_state, int target_layer, uint64_t* rng_state, uint32_t* rng_multiplier);

// Atomic operations for thread-safe detection accumulation
__device__ void atomic_add(uint64_t* memory_address, uint32_t value_to_add);

////////////////////////////////////////////////////////////////////////////////
// MAIN MONTE CARLO TRANSPORT KERNEL

/**
 * Primary Monte Carlo photon transport kernel
 *
 * This is the core computational kernel that simulates photon transport through
 * multi-layered biological tissues. Each GPU thread simulates one photon at a time,
 * with thousands of threads running in parallel for maximum computational throughput.
 *
 * COMPUTATIONAL WORKFLOW:
 * -----------------------
 * 1. Thread initialization: Load photon state and RNG state from global memory
 * 2. Main transport loop: Propagate photon through tissue layers
 *    - Sample step length from exponential distribution
 *    - Check for layer boundary crossings
 *    - Handle Fresnel reflection/transmission at boundaries
 *    - Accumulate absorbed energy and detection events
 *    - Perform anisotropic scattering (Henyey-Greenberg)
 *    - Apply Russian roulette termination for low-weight photons
 * 3. Photon lifecycle management: Launch new photons as current ones terminate
 * 4. State preservation: Save photon and RNG states to global memory
 *
 * TEMPLATE SPECIALIZATION:
 * ------------------------
 * The template parameter 'ignoreAdetection' enables compile-time optimization:
 * - ignoreAdetection=0: Full simulation with absorption detection (default)
 * - ignoreAdetection=1: Skip absorption detection for performance optimization
 *
 * PERFORMANCE CHARACTERISTICS:
 * ----------------------------
 * - Memory bandwidth optimized through coalesced access patterns
 * - Divergent branching minimized for SIMT efficiency
 * - Register usage optimized for maximum occupancy
 * - Atomic operations used only for thread-safe accumulation
 *
 * @param DeviceMem Complete device memory structure with all simulation arrays
 */
template<int ignoreAdetection>
__global__ void mc_d(MemStruct device_memory) {
	// THREAD IDENTIFICATION AND MEMORY ACCESS SETUP

	// Calculate global thread ID and memory access indices
	const int block_id = blockIdx.x;   // Block index within grid
	const int thread_id = threadIdx.x; // Thread index within block
	const int global_thread_index = NUM_THREADS_PER_BLOCK * block_id + thread_id;

	// Load RNG state and multipliers for this thread (coalesced access)
	uint64_t rng_state = device_memory.x[global_thread_index];      // MWC RNG state
	uint32_t rng_multiplier = device_memory.a[global_thread_index]; // Safe prime multiplier

	// Physics and detection variables
	float step_length;                     // Photon step length [cm]
	uint32_t detection_index;              // Array index for detection accumulation
	uint32_t absorbed_weight = 0;          // Accumulated absorbed weight
	uint32_t detection_index_previous = 0; // Previous detection index for optimization

	// Load photon state from global memory
	PhotonStruct photon = device_memory.p[global_thread_index];

	// Layer transition tracking
	int target_layer;

	// THREAD ACTIVITY CHECK AND MAIN TRANSPORT LOOP INITIALIZATION

	// Initialize loop counter and check thread activity status
	uint32_t loop_iteration = 0;
	if (!device_memory.thread_active[global_thread_index]) {
		loop_iteration = NUM_STEPS_GPU; // Skip main loop if thread is inactive
	}

	// MAIN MONTE CARLO TRANSPORT LOOP

	for (; loop_iteration < NUM_STEPS_GPU; loop_iteration++) {
		//=======================================================================
		// STEP LENGTH SAMPLING
		//=======================================================================

		// Sample step length from exponential distribution: s = -ln(ξ)/μₜ
		if (layers_dc[photon.layer].mutr != FLT_MAX) {
			// Normal tissue: exponential sampling with transport mean free path
			step_length = -__logf(rand_mwc_oc(&rng_state, &rng_multiplier)) * layers_dc[photon.layer].mutr;
		}
		else {
			// Glass layer: effectively infinite mean free path
			step_length = 100.0f; // Large step through non-scattering medium
		}

		// LAYER BOUNDARY INTERSECTION ANALYSIS

		// Initialize target layer (may change due to boundary crossing)
		target_layer = photon.layer;

		// Check for upward boundary crossing (reflection/transmission to layer above)
		if (photon.z + step_length * photon.dz < layers_dc[photon.layer].z_min) {
			target_layer = photon.layer - 1;
			step_length = __fdividef(layers_dc[photon.layer].z_min - photon.z, photon.dz);
		}

		// Check for downward boundary crossing (reflection/transmission to layer below)
		if (photon.z + step_length * photon.dz > layers_dc[photon.layer].z_max) {
			target_layer = photon.layer + 1;
			step_length = __fdividef(layers_dc[photon.layer].z_max - photon.z, photon.dz);
		}

		// PHOTON PROPAGATION

		// Move photon to new position
		photon.x += photon.dx * step_length;
		photon.y += photon.dy * step_length;
		photon.z += photon.dz * step_length;

		// Ensure photon stays within layer boundaries (numerical precision safety)
		photon.z = fminf(photon.z, layers_dc[photon.layer].z_max);
		photon.z = fmaxf(photon.z, layers_dc[photon.layer].z_min);

		// BOUNDARY INTERACTION PROCESSING

		if (target_layer != photon.layer) {
			// Reset step length for boundary interaction
			step_length = 0.0f;

			// Process Fresnel reflection/transmission at layer boundary
			if (reflect(&photon, target_layer, &rng_state, &rng_multiplier) == 0U) {
				// Photon transmitted through boundary

				// Diffuse reflectance detection (exiting top surface)
				if (target_layer == 0) {
					// Calculate detection bin indices: [angle][radius]
					const int angle_bin = __float2int_rz(acosf(-photon.dz) * 2.0f * RPI * det_dc[0].na);
					const int radius_bin =
						min(__float2int_rz(__fdividef(sqrtf(photon.x * photon.x + photon.y * photon.y), det_dc[0].dr)),
							static_cast<int>(det_dc[0].nr) - 1);
					detection_index = angle_bin * det_dc[0].nr + radius_bin;

					// Accumulate reflectance detection with thread-safe atomic operation
					atomic_add(&device_memory.Rd_ra[detection_index], photon.weight);

					// Terminate this photon (set weight to zero)
					photon.weight = 0;
				}

				// Transmittance detection (exiting bottom surface)
				if (target_layer > *n_layers_dc) {
					// Calculate detection bin indices: [angle][radius]
					const int angle_bin = __float2int_rz(acosf(photon.dz) * 2.0f * RPI * det_dc[0].na);
					const int radius_bin =
						min(__float2int_rz(__fdividef(sqrtf(photon.x * photon.x + photon.y * photon.y), det_dc[0].dr)),
							static_cast<int>(det_dc[0].nr) - 1);
					detection_index = angle_bin * det_dc[0].nr + radius_bin;

					// Accumulate transmittance detection with thread-safe atomic operation
					atomic_add(&device_memory.Tt_ra[detection_index], photon.weight);

					// Terminate this photon (set weight to zero)
					photon.weight = 0;
				}
			}
		}

		// ABSORPTION AND SCATTERING PROCESSING

		if (step_length > 0.0f) {
			// ENERGY ABSORPTION CALCULATION

			// Calculate absorbed weight using Beer-Lambert law: ΔW = W × μₐ/μₜ × s
			const uint32_t weight_absorbed = __float2uint_rn(layers_dc[photon.layer].mua * layers_dc[photon.layer].mutr
															 * __uint2float_rn(photon.weight));
			photon.weight -= weight_absorbed;

			// ABSORPTION DETECTION ACCUMULATION (TEMPLATE-CONTROLLED)

			// Template specialization: compile-time optimization for absorption detection
			if (ignoreAdetection == 0) {
				// Calculate detection bin index: A(r,z) spatial grid
				const int depth_bin =
					min(__float2int_rz(__fdividef(photon.z, det_dc[0].dz)), static_cast<int>(det_dc[0].nz) - 1);
				const int radius_bin =
					min(__float2int_rz(__fdividef(sqrtf(photon.x * photon.x + photon.y * photon.y), det_dc[0].dr)),
						static_cast<int>(det_dc[0].nr) - 1);
				detection_index = depth_bin * det_dc[0].nr + radius_bin;

				// Optimize atomic operations by accumulating within same spatial bin
				if (detection_index == detection_index_previous) {
					absorbed_weight += weight_absorbed;
				}
				else {
					// Commit previous accumulated weight to global memory
					if (absorbed_weight > 0) {
						atomic_add(&device_memory.A_rz[detection_index_previous], absorbed_weight);
					}
					// Start new accumulation for current spatial bin
					detection_index_previous = detection_index;
					absorbed_weight = weight_absorbed;
				}
			}

			// ANISOTROPIC SCATTERING (HENYEY-GREENBERG)

			// Apply anisotropic scattering with layer-specific anisotropy parameter
			spin(&photon, layers_dc[photon.layer].g, &rng_state, &rng_multiplier);
		}

		// PHOTON SURVIVAL AND LIFECYCLE MANAGEMENT

		// Check photon survival using Russian roulette termination
		if (!photon_survive(&photon, &rng_state, &rng_multiplier)) {
			// Current photon terminated - check if new photon should be launched
			if (atomicAdd(device_memory.num_terminated_photons, 1U) < (*num_photons_dc - NUM_THREADS)) {
				// Launch new photon to maintain simulation throughput
				launch_photon(&photon, &rng_state, &rng_multiplier);
			}
			else {
				// No more photons needed - deactivate this thread
				device_memory.thread_active[global_thread_index] = 0U;
				loop_iteration = NUM_STEPS_GPU; // Exit main transport loop
			}
		}

		// FINAL ABSORPTION ACCUMULATION (TEMPLATE-CONTROLLED CLEANUP)

		// Handle any remaining accumulated absorption weight
		if (ignoreAdetection == 1 && absorbed_weight != 0) {
			atomic_add(&device_memory.A_rz[detection_index_previous], absorbed_weight);
		}
	}

	// THREAD SYNCHRONIZATION AND STATE PRESERVATION

	// Ensure all threads complete before state saving (may not be necessary)
	__syncthreads();

	// Save photon and RNG states to global memory for potential continuation
	device_memory.p[global_thread_index] = photon;
	device_memory.x[global_thread_index] = rng_state;
}

////////////////////////////////////////////////////////////////////////////////
// PHOTON INITIALIZATION AND LIFECYCLE MANAGEMENT

/**
 * Initialize new photon state
 *
 * Sets up a new photon with standard initialization parameters for pencil
 * beam incident on the top surface of the multi-layered medium. This function
 * is called when a photon is first launched or when an old photon terminates
 * and needs to be replaced.
 *
 * INITIALIZATION PARAMETERS:
 * --------------------------
 * - Position: (0,0,0) - pencil beam incident at origin
 * - Direction: (0,0,1) - normal incidence (downward)
 * - Layer: 1 - first tissue layer (layer 0 is air above)
 * - Weight: Adjusted for specular reflection at top surface
 *
 * PHYSICAL CONSIDERATIONS:
 * ------------------------
 * - Specular reflection already accounted for in initial weight
 * - Fresnel reflection at air-tissue boundary handled during boundary crossing
 * - Weight normalization ensures proper statistical sampling
 *
 * @param photon_state   Pointer to photon structure to initialize
 * @param rng_state      Pointer to RNG state (not used in current implementation)
 * @param rng_multiplier Pointer to RNG multiplier (not used in current implementation)
 */
__device__ void launch_photon(PhotonStruct* photon_state, uint64_t* rng_state, uint32_t* rng_multiplier) {
	// Set initial position: pencil beam at origin
	photon_state->x = 0.0f; // [cm] - lateral position x
	photon_state->y = 0.0f; // [cm] - lateral position y
	photon_state->z = 0.0f; // [cm] - depth position (top surface)

	// Set initial direction: normal incidence (downward)
	photon_state->dx = 0.0f; // x-direction cosine
	photon_state->dy = 0.0f; // y-direction cosine
	photon_state->dz = 1.0f; // z-direction cosine (downward)

	// Set initial layer: first tissue layer (not air above)
	photon_state->layer = 1;

	// Set initial weight: accounts for specular reflection at top surface
	photon_state->weight = *start_weight_dc;
}

/**
 * Global photon launch kernel
 *
 * GPU kernel for initializing photon states across all threads simultaneously.
 * This kernel is typically called once at the beginning of simulation to
 * set up the initial photon population across all GPU threads.
 *
 * KERNEL EXECUTION MODEL:
 * -----------------------
 * - One thread per photon initialization
 * - Coalesced memory access for optimal bandwidth
 * - Minimal computation - just initialization
 *
 * @param device_memory Complete device memory structure containing photon arrays
 */
__global__ void launch_photon_global(MemStruct device_memory) {
	// Calculate global thread index for memory access
	const int block_id = blockIdx.x;
	const int thread_id = threadIdx.x;
	const int global_thread_index = NUM_THREADS_PER_BLOCK * block_id + thread_id;

	// Local photon structure for initialization
	PhotonStruct photon;

	// Load RNG state for this thread (though not used in current launch_photon)
	uint64_t rng_state = device_memory.x[global_thread_index];
	uint32_t rng_multiplier = device_memory.a[global_thread_index];

	// Initialize photon with standard parameters
	launch_photon(&photon, &rng_state, &rng_multiplier);

	// Store initialized photon state to global memory
	device_memory.p[global_thread_index] = photon;
}

////////////////////////////////////////////////////////////////////////////////
// ANISOTROPIC SCATTERING IMPLEMENTATION

/**
 * Henyey-Greenberg anisotropic scattering
 *
 * Updates photon direction according to the Henyey-Greenberg phase function,
 * which models anisotropic scattering in biological tissues. The scattering
 * anisotropy parameter g controls the forward/backward scattering preference.
 *
 * PHASE FUNCTION MATHEMATICS:
 * ---------------------------
 * P(cos θ) = (1-g²) / (1 + g² - 2g cos θ)^(3/2)
 *
 * Where θ is the scattering angle and g is the anisotropy parameter:
 * - g = 0: Isotropic scattering (uniform in all directions)
 * - g > 0: Forward-peaked scattering (typical for biological tissues)
 * - g < 0: Back-scattered scattering (rare in biology)
 * - |g| → 1: Highly anisotropic scattering
 *
 * SCATTERING COORDINATE SYSTEM:
 * ------------------------------
 * - Polar angle θ: Sampled from Henyey-Greenberg distribution
 * - Azimuthal angle φ: Uniformly distributed [0, 2π)
 * - Coordinate transformation from local to global coordinate system
 * - Special handling for normal incidence (dz ≈ ±1)
 *
 * NUMERICAL OPTIMIZATION:
 * -----------------------
 * - GPU intrinsic functions for fast trigonometry
 * - Optimized coordinate transformations
 * - Renormalization to handle floating-point precision loss
 *
 * @param photon_state   Pointer to photon structure (direction modified in-place)
 * @param anisotropy_g   Anisotropy parameter [-1,1] for Henyey-Greenberg scattering
 * @param rng_state      Pointer to RNG state
 * @param rng_multiplier Pointer to RNG multiplier
 */
__device__ void spin(PhotonStruct* photon_state, float anisotropy_g, uint64_t* rng_state, uint32_t* rng_multiplier) {
	// Scattering angle parameters
	float cos_theta, sin_theta; // Cosine and sine of polar scattering angle θ
	float cos_phi, sin_phi;     // Cosine and sine of azimuthal angle φ
	float temp, temp_dx;        // Temporary variables for coordinate transformation

	// POLAR ANGLE SAMPLING (HENYEY-GREENBERG)

	if (anisotropy_g == 0.0f) {
		// Special case: Isotropic scattering (g = 0)
		cos_theta = 2.0f * rand_mwc_co(rng_state, rng_multiplier) - 1.0f; // Uniform [-1, 1]
	}
	else {
		// General case: Anisotropic Henyey-Greenberg scattering
		// Sample using inverse transform method
		temp = __fdividef((1.0f - anisotropy_g * anisotropy_g), 
						  (1.0f - anisotropy_g + 2.0f * anisotropy_g * rand_mwc_co(rng_state, rng_multiplier)));
		cos_theta = __fdividef((1.0f + anisotropy_g * anisotropy_g - temp * temp), (2.0f * anisotropy_g));
	}

	// Calculate sine from cosine (sin² + cos² = 1)
	sin_theta = sqrtf(1.0f - cos_theta * cos_theta);

	// AZIMUTHAL ANGLE SAMPLING

	// Sample azimuthal angle uniformly from [0, 2π)
	__sincosf(2.0f * PI * rand_mwc_co(rng_state, rng_multiplier), &sin_phi, &cos_phi);

	// COORDINATE SYSTEM TRANSFORMATION

	// Calculate perpendicular component magnitude for coordinate transformation
	temp = sqrtf(1.0f - photon_state->dz * photon_state->dz);

	if (temp == 0.0f) {
		// SPECIAL CASE: NORMAL INCIDENCE (dz ≈ ±1)

		// Simple transformation for normal incidence
		photon_state->dx = sin_theta * cos_phi;
		photon_state->dy = sin_theta * sin_phi;
		photon_state->dz = copysignf(cos_theta, photon_state->dz * cos_theta); // Preserve sign
	}
	else {
		// GENERAL CASE: OBLIQUE INCIDENCE

		// Store original dx for coordinate transformation
		temp_dx = photon_state->dx;

		// Apply full 3D coordinate transformation
		photon_state->dx = __fdividef(sin_theta * (photon_state->dx * photon_state->dz * cos_phi - photon_state->dy * sin_phi), temp) + photon_state->dx * cos_theta;
		photon_state->dy = __fdividef(sin_theta * (photon_state->dy * photon_state->dz * cos_phi + temp_dx * sin_phi), temp) + photon_state->dy * cos_theta;
		photon_state->dz = -sin_theta * cos_phi * temp + photon_state->dz * cos_theta;
	}

	// DIRECTION VECTOR RENORMALIZATION

	// Renormalize direction vector to account for floating-point precision loss
	// This is critical for maintaining unit vector properties over many scattering events
	temp = rsqrtf(photon_state->dx * photon_state->dx + photon_state->dy * photon_state->dy + photon_state->dz * photon_state->dz); // Fast inverse square root
	photon_state->dx *= temp;
	photon_state->dy *= temp;
	photon_state->dz *= temp;
}

////////////////////////////////////////////////////////////////////////////////
// FRESNEL REFLECTION AND TRANSMISSION

/**
 * Process Fresnel reflection/transmission at layer boundaries
 *
 * Calculates whether a photon is reflected or transmitted when crossing
 * a boundary between layers with different refractive indices. Uses the
 * complete Fresnel equations to account for both polarization states
 * and handles special cases like total internal reflection.
 *
 * FRESNEL REFLECTION PHYSICS:
 * ---------------------------
 * The reflection probability depends on:
 * - Incident angle θᵢ relative to surface normal
 * - Refractive indices of both layers (n₁, n₂)
 * - Polarization state (averaged over both s and p polarizations)
 *
 * SPECIAL CASES:
 * --------------
 * 1. Refractive index matching (n₁ = n₂): Automatic transmission
 * 2. Normal incidence (θᵢ = 0): Simple Fresnel formula
 * 3. Total internal reflection (n₁ > n₂, θᵢ > θc): Complete reflection
 * 4. General case: Full Fresnel calculation with optimized algorithm
 *
 * COMPUTATIONAL OPTIMIZATIONS:
 * -----------------------------
 * - Fast algorithm avoids expensive inverse trigonometry
 * - GPU-optimized mathematical operations
 * - Single random number sample for reflection/transmission decision
 * - Efficient direction vector updates
 *
 * @param p         Pointer to photon structure (layer and direction modified)
 * @param new_layer Target layer index if transmitted
 * @param x         Pointer to RNG state
 * @param a         Pointer to RNG multiplier
 * @return 1 if photon is reflected, 0 if transmitted
 */
__device__ uint32_t reflect(PhotonStruct* photon_state, int target_layer, uint64_t* rng_state, uint32_t* rng_multiplier) {
	// REFRACTIVE INDICES AND INITIAL SETUP

	// Extract refractive indices for current and target layers
	const float n1 = layers_dc[photon_state->layer].n;  // Current layer refractive index
	const float n2 = layers_dc[target_layer].n; // Target layer refractive index

	// Calculate incident angle cosine (angle with respect to surface normal)
	const float cos_incident = fabsf(photon_state->dz);

	// SPECIAL CASE: REFRACTIVE INDEX MATCHING

	if (n1 == n2) {
		// Perfect index matching: automatic transmission with no direction change
		photon_state->layer = target_layer;
		return 0U; // Transmitted
	}

	// SPECIAL CASE: TOTAL INTERNAL REFLECTION

	if (n1 > n2) {
		// Check for total internal reflection condition
		// Critical angle: sin(θc) = n₂/n₁
		// TIR occurs when: sin²(θᵢ) > (n₂/n₁)²
		const float sin_squared_incident = 1.0f - cos_incident * cos_incident;
		const float index_ratio_squared = (n2 / n1) * (n2 / n1);

		if (sin_squared_incident > index_ratio_squared) {
			// Total internal reflection: mirror z-direction only
			photon_state->dz *= -1.0f;
			return 1U; // Reflected
		}
	}

	// SPECIAL CASE: NORMAL INCIDENCE

	if (cos_incident == 1.0f) {
		// Normal incidence: simplified Fresnel formula
		const float r_normal = (n1 - n2) / (n1 + n2);
		const float reflectance = r_normal * r_normal;

		if (rand_mwc_co(rng_state, rng_multiplier) <= reflectance) {
			// Reflection: mirror z-direction only
			photon_state->dz *= -1.0f;
			return 1U; // Reflected
		}
		else {
			// Transmission: no direction change, only layer update
			photon_state->layer = target_layer;
			return 0U; // Transmitted
		}
	}

	// GENERAL CASE: FULL FRESNEL CALCULATION

	// Calculate transmission angle using Snell's law
	// sin²(θₜ) = (n₁/n₂)² × sin²(θᵢ)
	const float index_ratio = n1 / n2;
	const float sin_squared_transmitted = index_ratio * index_ratio * (1.0f - cos_incident * cos_incident);

	// Optimized Fresnel reflectance calculation
	// This algorithm avoids expensive trigonometric functions while maintaining accuracy
	float temp_factor = 2.0f
						* sqrtf((1.0f - cos_incident * cos_incident) * (1.0f - sin_squared_transmitted)
								* sin_squared_transmitted * cos_incident * cos_incident);

	// Intermediate calculation for Fresnel coefficients
	float fresnel_term =
		sin_squared_transmitted + (cos_incident * cos_incident) * (1.0f - 2.0f * sin_squared_transmitted);

	// Calculate final reflectance (averaged over both polarizations)
	float reflectance = fresnel_term
						* __fdividef((1.0f - fresnel_term - temp_factor),
									 ((1.0f - fresnel_term + temp_factor) * (fresnel_term + temp_factor)));

	// REFLECTION/TRANSMISSION DECISION

	if (rand_mwc_co(rng_state, rng_multiplier) <= reflectance) {
		// PHOTON REFLECTED
		photon_state->dz *= -1.0f; // Mirror z-direction component
		return 1U;      // Reflected
	}
	else {
		// PHOTON TRANSMITTED

		// Update direction vector for refracted ray
		const float cos_transmitted = sqrtf(1.0f - sin_squared_transmitted);

		// Scale lateral components by index ratio
		photon_state->dx *= index_ratio;
		photon_state->dy *= index_ratio;

		// Calculate transmitted z-component (preserve sign, use refracted magnitude)
		photon_state->dz = copysignf(cos_transmitted, photon_state->dz);

		// Update photon layer
		photon_state->layer = target_layer;

		return 0U; // Transmitted
	}
}

////////////////////////////////////////////////////////////////////////////////
// PHOTON SURVIVAL AND TERMINATION

/**
 * Russian roulette photon survival algorithm
 *
 * Determines whether a low-weight photon survives for continued simulation
 * or is terminated to maintain computational efficiency. Uses the Russian
 * roulette technique to maintain statistical accuracy while eliminating
 * photons that contribute negligibly to the final result.
 *
 * RUSSIAN ROULETTE ALGORITHM:
 * ---------------------------
 * - High-weight photons (W > WEIGHTI): Always survive
 * - Zero-weight photons: Always terminate (exited simulation domain)
 * - Low-weight photons: Survive with probability CHANCE
 * - Surviving photons: Weight boosted by factor 1/CHANCE
 *
 * STATISTICAL CORRECTNESS:
 * ------------------------
 * The expected contribution remains unchanged:
 * E[W_new] = CHANCE × (W/CHANCE) + (1-CHANCE) × 0 = W
 *
 * This maintains unbiased results while eliminating computationally
 * expensive tracking of very low-weight photons.
 *
 * @param p Pointer to photon structure (weight may be modified)
 * @param x Pointer to RNG state
 * @param a Pointer to RNG multiplier
 * @return 1 if photon survives, 0 if terminated
 */
__device__ uint32_t photon_survive(PhotonStruct* photon_state, uint64_t* rng_state, uint32_t* rng_multiplier) {
	// HIGH-WEIGHT PHOTON: AUTOMATIC SURVIVAL

	if (photon_state->weight > WEIGHTI) {
		return 1U;  // High-weight photon always survives
	}

	// ZERO-WEIGHT PHOTON: AUTOMATIC TERMINATION

	if (photon_state->weight == 0U) {
		return 0U; // Zero-weight photon (exited domain) always terminates
	}

	// LOW-WEIGHT PHOTON: RUSSIAN ROULETTE DECISION

	// Apply Russian roulette with survival probability CHANCE
	if (rand_mwc_co(rng_state, rng_multiplier) < CHANCE) {
		// Photon survives: boost weight to maintain statistical accuracy
		photon_state->weight = __float2uint_rn(__fdividef(static_cast<float>(photon_state->weight), CHANCE));
		return 1U; // Survived with boosted weight
	}

	// Photon terminated by Russian roulette
	return 0U;
}

////////////////////////////////////////////////////////////////////////////////
// ATOMIC OPERATIONS FOR THREAD-SAFE DETECTION

/**
 * 64-bit atomic addition for detection accumulation
 *
 * Performs thread-safe addition of 32-bit values to 64-bit detection arrays.
 * This function is essential for accumulating detection events from multiple
 * GPU threads without race conditions or data corruption.
 *
 * IMPLEMENTATION DETAILS:
 * -----------------------
 * - Uses 32-bit atomic operations on lower and upper halves separately
 * - Handles carry propagation between lower and upper 32-bit words
 * - Compatible with GPU Compute Capability 1.1+ requirements
 * - Ensures atomicity of complete 64-bit operation
 *
 * PERFORMANCE CONSIDERATIONS:
 * ---------------------------
 * - Optimized for common case (no carry propagation)
 * - Minimal overhead for most addition operations
 * - Coalesced memory access when possible
 * - Essential for massively parallel detection accumulation
 *
 * @param address Pointer to 64-bit value to be incremented
 * @param add     32-bit value to add atomically
 */
__device__ void atomic_add(uint64_t* memory_address, uint32_t value_to_add) {
	// Perform atomic addition on lower 32 bits
	uint32_t old_lower = atomicAdd(reinterpret_cast<uint32_t*>(memory_address), value_to_add);

	// Check for overflow (carry needed to upper 32 bits)
	// Overflow condition: (old_lower + value_to_add) < value_to_add, which indicates wraparound
	if (old_lower + value_to_add < value_to_add) {
		// Propagate carry to upper 32 bits
		atomicAdd(reinterpret_cast<uint32_t*>(memory_address) + 1, 1U);
	}
}
