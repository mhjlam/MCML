/*==============================================================================
 * CUDAMCML Memory Module - GPU Memory Management for Multi-Layer Simulations
 *
 * This module provides comprehensive memory management functionality for
 * CUDAMCML simulations, including host/device allocation, data transfers,
 * and cleanup operations. Implements modern C++ RAII principles with
 * robust error handling for GPU memory operations.
 *
 * MEMORY ARCHITECTURE:
 * --------------------
 * - Host Memory: Standard RAM for CPU-side data processing
 * - Device Memory: GPU global memory for parallel computation
 * - Constant Memory: GPU constant cache for frequently accessed parameters
 * - Shared Memory: GPU on-chip memory for thread block communication
 *
 * ALLOCATION STRATEGY:
 * --------------------
 * - Detection Arrays: Large 2D grids for spatial/angular detection
 * - Photon States: Individual photon tracking structures per GPU thread
 * - RNG States: Random number generator state arrays for statistical quality
 * - Control Structures: Thread management and synchronization primitives
 *
 * ERROR HANDLING:
 * ---------------
 * - Comprehensive CUDA error checking with detailed diagnostics
 * - Memory leak prevention with proper cleanup routines
 * - Graceful degradation on allocation failures
 * - Resource tracking for debugging and optimization
 *
 * LICENSE:
 * --------
 * This file is part of CUDAMCML.
 *
 * CUDAMCML is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * CUDAMCML is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with CUDAMCML.  If not, see <http://www.gnu.org/licenses/>.
 */

////////////////////////////////////////////////////////////////////////////////
// INCLUDES AND DEPENDENCIES

// Project-specific headers
#include "cudamcml.h"

// Standard library includes
#include <cstdint> // Standard integer types for cross-platform compatibility
#include <cstdio>  // C-style I/O for error reporting
#include <cstdlib> // Memory allocation and process control

////////////////////////////////////////////////////////////////////////////////
// DEVICE-TO-HOST DATA TRANSFER OPERATIONS

/**
 * Transfer simulation results from GPU to host memory
 *
 * Copies all detection arrays and RNG states from GPU device memory
 * back to host memory for analysis and output generation. This is the
 * final step in the GPU computation pipeline.
 *
 * TRANSFER OPERATIONS:
 * --------------------
 * 1. Absorption data: A(r,z) - volumetric energy deposition
 * 2. Reflectance data: Rd(r,α) - top surface detection with angular resolution
 * 3. Transmittance data: Tt(r,α) - bottom surface detection with angular resolution
 * 4. RNG states: Preserved for simulation continuation or analysis
 *
 * PERFORMANCE CONSIDERATIONS:
 * ---------------------------
 * - Uses asynchronous transfers where possible for overlap
 * - Memory access patterns optimized for coalescing
 * - Transfer sizes calculated to minimize GPU memory bandwidth usage
 *
 * @param HostMem    Pointer to host memory structure (destination)
 * @param DeviceMem  Pointer to device memory structure (source)
 * @param sim        Pointer to simulation configuration for array sizing
 *
 * @return 0 on success, non-zero on CUDA transfer errors
 */
auto copy_device_to_host_mem(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim) -> int {
	// Calculate detection array sizes based on grid configuration
	const int rz_size = sim->det.nr * sim->det.nz; // Absorption array size [r * z]
	const int ra_size = sim->det.nr * sim->det.na; // Reflectance/transmittance array size [r * a]

	printf("Transferring simulation results from GPU to host memory...\n");
	printf("  - Absorption array: %d * %d = %d elements\n", sim->det.nr, sim->det.nz, rz_size);
	printf("  - Reflectance/transmittance arrays: %d * %d = %d elements each\n", sim->det.nr, sim->det.na, ra_size);

	// DETECTION DATA TRANSFERS

	// Transfer absorption detection array A(r,z)
	CUDA_CHECK_ERROR(hipMemcpy(HostMem->A_rz, DeviceMem->A_rz, rz_size * sizeof(uint64_t), hipMemcpyDeviceToHost));

	// Transfer reflectance detection array Rd(r,α)
	CUDA_CHECK_ERROR(hipMemcpy(HostMem->Rd_ra, DeviceMem->Rd_ra, ra_size * sizeof(uint64_t), hipMemcpyDeviceToHost));

	// Transfer transmittance detection array Tt(r,α)
	CUDA_CHECK_ERROR(hipMemcpy(HostMem->Tt_ra, DeviceMem->Tt_ra, ra_size * sizeof(uint64_t), hipMemcpyDeviceToHost));

	// RANDOM NUMBER GENERATOR STATE PRESERVATION

	// Transfer RNG states for potential simulation continuation
	// This allows for reproducible results and debugging capabilities
	CUDA_CHECK_ERROR(hipMemcpy(HostMem->x, DeviceMem->x, NUM_THREADS * sizeof(uint64_t), hipMemcpyDeviceToHost));

	printf("GPU-to-host transfer completed successfully.\n");
	return 0;
}

////////////////////////////////////////////////////////////////////////////////
// CONSTANT MEMORY INITIALIZATION

/**
 * Initialize GPU constant memory with simulation parameters
 *
 * Transfers frequently-accessed simulation parameters to GPU constant
 * memory for optimal device kernel performance. Constant memory is
 * cached and broadcast efficiently to all threads simultaneously.
 *
 * CONSTANT MEMORY CONTENTS:
 * -------------------------
 * - Detection grid configuration (DetStruct)
 * - Layer count and optical properties (LayerStruct array)
 * - Photon weight initialization parameters
 * - Total photon count for normalization
 *
 * PERFORMANCE BENEFITS:
 * ---------------------
 * - Single read broadcasts to all threads in a warp
 * - Cached for repeated access patterns
 * - Eliminates global memory traffic for parameters
 * - Reduces register pressure in device kernels
 *
 * @param sim Pointer to simulation configuration structure
 * @return 0 on success, non-zero on CUDA transfer errors
 */
auto init_dc_mem(SimulationStruct* sim) -> int {
	printf("Initializing GPU constant memory with simulation parameters...\n");

	// Transfer detection grid configuration to constant memory
	CUDA_CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(det_dc), &(sim->det), sizeof(DetStruct)));
	printf("  - Detection grid: %d*%d*%d (r*z*a)\n", sim->det.nr, sim->det.nz, sim->det.na);

	// Transfer layer count for boundary checking
	CUDA_CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(n_layers_dc), &(sim->n_layers), sizeof(uint32_t)));
	printf("  - Number of tissue layers: %u\n", sim->n_layers);

	// Transfer initial photon weight for normalization
	CUDA_CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(start_weight_dc), &(sim->start_weight), sizeof(uint32_t)));
	printf("  - Initial photon weight: 0x%08X\n", sim->start_weight);

	// Transfer complete layer optical properties array
	// Include boundary layers (air above and below) for Fresnel calculations
	const size_t layer_array_size = (sim->n_layers + 2) * sizeof(LayerStruct);
	CUDA_CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(layers_dc), sim->layers, layer_array_size));
	printf("  - Layer properties: %zu bytes (%u layers + 2 boundaries)\n", layer_array_size, sim->n_layers);

	// Transfer total photon count for statistical normalization
	CUDA_CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(num_photons_dc), &(sim->number_of_photons), sizeof(uint32_t)));
	printf("  - Total photon count: %lu\n", sim->number_of_photons);

	printf("Constant memory initialization completed successfully.\n");
	return 0;
}

////////////////////////////////////////////////////////////////////////////////
// COMPREHENSIVE MEMORY ALLOCATION AND INITIALIZATION

/**
 * Initialize all host and device memory structures
 *
 * Performs comprehensive memory allocation for both CPU and GPU memory
 * structures required for Monte Carlo simulation. Implements proper
 * error checking and initialization to ensure reliable operation.
 *
 * ALLOCATION STRATEGY:
 * --------------------
 * 1. Detection Arrays: Large 2D grids for spatial/angular data collection
 * 2. Photon State Arrays: Individual tracking structures per GPU thread
 * 3. RNG State Management: High-quality random number generator states
 * 4. Thread Control: Synchronization and status tracking arrays
 * 5. Performance Counters: Simulation progress and statistics tracking
 *
 * MEMORY ORGANIZATION:
 * --------------------
 * - Host Memory: CPU-accessible for I/O and post-processing
 * - Device Memory: GPU-optimized for parallel computation
 * - Initialization: Proper zero-filling and state setup
 * - Error Handling: Comprehensive validation and cleanup
 *
 * @param HostMem    Pointer to host memory structure to initialize
 * @param DeviceMem  Pointer to device memory structure to initialize
 * @param sim        Pointer to simulation configuration
 * @return 1 on success, 0 on allocation failure
 */
auto init_mem_structs(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim) -> int {
	// Calculate detection array sizes based on simulation configuration
	const int rz_size = sim->det.nr * sim->det.nz; // Absorption detection array size
	const int ra_size = sim->det.nr * sim->det.na; // Reflectance/transmittance array size

	printf("Initializing memory structures for CUDAMCML simulation...\n");
	printf("Memory allocation summary:\n");
	printf("  - GPU threads: %d\n", NUM_THREADS);
	printf("  - Detection grid (r*z): %d*%d = %d elements\n", sim->det.nr, sim->det.nz, rz_size);
	printf("  - Detection grid (r*a): %d*%d = %d elements\n", sim->det.nr, sim->det.na, ra_size);

	// PHOTON STATE ARRAY ALLOCATION (GPU ONLY)

	// Allocate photon state structures on GPU device
	// Each thread maintains its own photon state during simulation
	CUDA_CHECK_ERROR(hipMalloc(reinterpret_cast<void**>(&DeviceMem->p), NUM_THREADS * sizeof(PhotonStruct)));
	printf("  Photon states: %zu MB (GPU)\n", (NUM_THREADS * sizeof(PhotonStruct)) / (1024 * 1024));

	// ABSORPTION DETECTION ARRAY ALLOCATION

	// Host allocation for absorption array A(r,z)
	HostMem->A_rz = static_cast<uint64_t*>(malloc(rz_size * sizeof(uint64_t)));
	if (HostMem->A_rz == nullptr) {
		fprintf(stderr, "Error: Failed to allocate host absorption array (%zu MB)\n",
				(rz_size * sizeof(uint64_t)) / (1024 * 1024));
		return 0;
	}

	// Device allocation and initialization for absorption array
	CUDA_CHECK_ERROR(hipMalloc(reinterpret_cast<void**>(&DeviceMem->A_rz), rz_size * sizeof(uint64_t)));
	CUDA_CHECK_ERROR(hipMemset(DeviceMem->A_rz, 0, rz_size * sizeof(uint64_t)));
	printf("  Absorption arrays: %zu MB (Host + GPU)\n", 2 * (rz_size * sizeof(uint64_t)) / (1024 * 1024));

	// REFLECTANCE DETECTION ARRAY ALLOCATION

	// Host allocation for reflectance array Rd(r,α)
	HostMem->Rd_ra = static_cast<uint64_t*>(malloc(ra_size * sizeof(uint64_t)));
	if (HostMem->Rd_ra == nullptr) {
		fprintf(stderr, "Error: Failed to allocate host reflectance array (%zu MB)\n",
				(ra_size * sizeof(uint64_t)) / (1024 * 1024));
		return 0;
	}

	// Device allocation and initialization for reflectance array
	CUDA_CHECK_ERROR(hipMalloc(reinterpret_cast<void**>(&DeviceMem->Rd_ra), ra_size * sizeof(uint64_t)));
	CUDA_CHECK_ERROR(hipMemset(DeviceMem->Rd_ra, 0, ra_size * sizeof(uint64_t)));
	printf("  Reflectance arrays: %zu MB (Host + GPU)\n", 2 * (ra_size * sizeof(uint64_t)) / (1024 * 1024));

	// TRANSMITTANCE DETECTION ARRAY ALLOCATION

	// Host allocation for transmittance array Tt(r,α)
	HostMem->Tt_ra = static_cast<uint64_t*>(malloc(ra_size * sizeof(uint64_t)));
	if (HostMem->Tt_ra == nullptr) {
		fprintf(stderr, "Error: Failed to allocate host transmittance array (%zu MB)\n",
				(ra_size * sizeof(uint64_t)) / (1024 * 1024));
		return 0;
	}

	// Device allocation and initialization for transmittance array
	CUDA_CHECK_ERROR(hipMalloc(reinterpret_cast<void**>(&DeviceMem->Tt_ra), ra_size * sizeof(uint64_t)));
	CUDA_CHECK_ERROR(hipMemset(DeviceMem->Tt_ra, 0, ra_size * sizeof(uint64_t)));
	printf("  Transmittance arrays: %zu MB (Host + GPU)\n", 2 * (ra_size * sizeof(uint64_t)) / (1024 * 1024));

	// RANDOM NUMBER GENERATOR STATE MANAGEMENT

	// Device allocation for RNG state arrays (x and a)
	// These contain the state and multipliers for high-quality MWC generators
	CUDA_CHECK_ERROR(hipMalloc(reinterpret_cast<void**>(&DeviceMem->x), NUM_THREADS * sizeof(uint64_t)));
	CUDA_CHECK_ERROR(hipMemcpy(DeviceMem->x, HostMem->x, NUM_THREADS * sizeof(uint64_t), hipMemcpyHostToDevice));

	CUDA_CHECK_ERROR(hipMalloc(reinterpret_cast<void**>(&DeviceMem->a), NUM_THREADS * sizeof(uint32_t)));
	CUDA_CHECK_ERROR(hipMemcpy(DeviceMem->a, HostMem->a, NUM_THREADS * sizeof(uint32_t), hipMemcpyHostToDevice));
	printf("  RNG states: %zu MB (Host + GPU)\n",
		   2 * (NUM_THREADS * (sizeof(uint64_t) + sizeof(uint32_t))) / (1024 * 1024));

	// THREAD ACTIVITY TRACKING

	// Host allocation and initialization for thread activity tracking
	HostMem->thread_active = static_cast<uint32_t*>(malloc(NUM_THREADS * sizeof(uint32_t)));
	if (HostMem->thread_active == nullptr) {
		fprintf(stderr, "Error: Failed to allocate thread activity array\n");
		return 0;
	}

	// Initialize all threads as active
	for (int i = 0; i < NUM_THREADS; i++) {
		HostMem->thread_active[i] = 1U;
	}

	// Device allocation and transfer for thread activity tracking
	CUDA_CHECK_ERROR(hipMalloc(reinterpret_cast<void**>(&DeviceMem->thread_active), NUM_THREADS * sizeof(uint32_t)));
	CUDA_CHECK_ERROR(hipMemcpy(DeviceMem->thread_active, HostMem->thread_active, NUM_THREADS * sizeof(uint32_t),
								hipMemcpyHostToDevice));

	// SIMULATION PROGRESS TRACKING

	// Host allocation and initialization for photon termination counter
	HostMem->num_terminated_photons = static_cast<uint32_t*>(malloc(sizeof(uint32_t)));
	if (HostMem->num_terminated_photons == nullptr) {
		fprintf(stderr, "Error: Failed to allocate termination counter\n");
		return 0;
	}

	*HostMem->num_terminated_photons = 0; // Initialize counter to zero

	// Device allocation and transfer for termination counter
	CUDA_CHECK_ERROR(hipMalloc(reinterpret_cast<void**>(&DeviceMem->num_terminated_photons), sizeof(uint32_t)));
	CUDA_CHECK_ERROR(hipMemcpy(DeviceMem->num_terminated_photons, HostMem->num_terminated_photons, sizeof(uint32_t),
								hipMemcpyHostToDevice));

	printf("  ✓ Control structures: Thread tracking and progress counters\n");
	printf("Memory initialization completed successfully.\n");

	return 1; // Success
}

////////////////////////////////////////////////////////////////////////////////
// MEMORY CLEANUP AND DEALLOCATION

/**
 * Comprehensive memory cleanup for host and device structures
 *
 * Performs proper deallocation of all memory structures allocated during
 * simulation initialization. Ensures no memory leaks and proper resource
 * cleanup for both host and device memory allocations.
 *
 * CLEANUP STRATEGY:
 * -----------------
 * - Host Memory: Standard free() calls for CPU-allocated arrays
 * - Device Memory: hipFree() calls for GPU-allocated arrays
 * - Error Resilience: Continues cleanup even if individual calls fail
 * - Resource Tracking: Ensures all allocated structures are deallocated
 *
 * @param HostMem    Pointer to host memory structure to deallocate
 * @param DeviceMem  Pointer to device memory structure to deallocate
 */
auto free_mem_structs(MemStruct* HostMem, MemStruct* DeviceMem) -> void {
	printf("Cleaning up memory structures...\n");

	//===========================================================================
	// HOST MEMORY DEALLOCATION
	//===========================================================================

	// Free host detection arrays
	if (HostMem->A_rz != nullptr) {
		free(HostMem->A_rz);
		HostMem->A_rz = nullptr;
	}

	if (HostMem->Rd_ra != nullptr) {
		free(HostMem->Rd_ra);
		HostMem->Rd_ra = nullptr;
	}

	if (HostMem->Tt_ra != nullptr) {
		free(HostMem->Tt_ra);
		HostMem->Tt_ra = nullptr;
	}

	// Free host control structures
	if (HostMem->thread_active != nullptr) {
		free(HostMem->thread_active);
		HostMem->thread_active = nullptr;
	}

	if (HostMem->num_terminated_photons != nullptr) {
		free(HostMem->num_terminated_photons);
		HostMem->num_terminated_photons = nullptr;
	}

	// DEVICE MEMORY DEALLOCATION

	// Free device detection arrays
	if (DeviceMem->A_rz != nullptr) {
		hipFree(DeviceMem->A_rz);
		DeviceMem->A_rz = nullptr;
	}

	if (DeviceMem->Rd_ra != nullptr) {
		hipFree(DeviceMem->Rd_ra);
		DeviceMem->Rd_ra = nullptr;
	}

	if (DeviceMem->Tt_ra != nullptr) {
		hipFree(DeviceMem->Tt_ra);
		DeviceMem->Tt_ra = nullptr;
	}

	// Free device RNG and photon state arrays
	if (DeviceMem->x != nullptr) {
		hipFree(DeviceMem->x);
		DeviceMem->x = nullptr;
	}

	if (DeviceMem->a != nullptr) {
		hipFree(DeviceMem->a);
		DeviceMem->a = nullptr;
	}

	if (DeviceMem->p != nullptr) {
		hipFree(DeviceMem->p);
		DeviceMem->p = nullptr;
	}

	// Free device control structures
	if (DeviceMem->thread_active != nullptr) {
		hipFree(DeviceMem->thread_active);
		DeviceMem->thread_active = nullptr;
	}

	if (DeviceMem->num_terminated_photons != nullptr) {
		hipFree(DeviceMem->num_terminated_photons);
		DeviceMem->num_terminated_photons = nullptr;
	}

	printf("Memory cleanup completed successfully.\n");
}

/**
 * Free simulation configuration structures
 *
 * Deallocates memory used by simulation configuration arrays, including
 * dynamically allocated layer property arrays for each simulation run.
 *
 * CLEANUP RESPONSIBILITIES:
 * -------------------------
 * - Layer property arrays for each simulation
 * - Main simulation structure array
 * - Proper handling of multiple simulation runs
 *
 * @param sim            Pointer to simulation structure array
 * @param n_simulations  Number of simulation configurations to clean up
 */
auto free_simulation_struct(SimulationStruct* sim, int n_simulations) -> void {
	if (sim == nullptr)
		return;

	printf("Cleaning up simulation configuration structures...\n");

	// Free layer arrays for each simulation
	for (int i = 0; i < n_simulations; i++) {
		if (sim[i].layers != nullptr) {
			free(sim[i].layers);
			sim[i].layers = nullptr;
		}
	}

	// Free main simulation array
	free(sim);
	printf("Simulation structure cleanup completed.\n");
}
