#include "hip/hip_runtime.h"
/*==============================================================================
 * CUDAMCML I/O Module - Input/Output Operations for Multi-Layer Simulations
 *
 * This module provides comprehensive I/O functionality for CUDAMCML simulations,
 * including parameter file parsing, result writing, and command-line argument
 * processing. Designed for compatibility with standard MCML file formats while
 * supporting GPU-specific optimizations.
 *
 * FUNCTIONALITY:
 * --------------
 * - Command-line argument parsing with validation
 * - Input file parsing for multi-layer tissue parameters
 * - Binary and ASCII output writing with proper formatting
 * - Error handling and validation for all I/O operations
 * - Memory-efficient file operations for large datasets
 *
 * FILE FORMAT COMPATIBILITY:
 * -------------------------
 * - Supports standard MCML input file format
 * - Compatible with existing MCML analysis tools
 * - Extensible for future format enhancements
 * - Binary output for high-precision results
 *
 * LICENSE:
 * --------
 * This file is part of CUDAMCML.
 *
 * CUDAMCML is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * CUDAMCML is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with CUDAMCML.  If not, see <http://www.gnu.org/licenses/>.
 */

////////////////////////////////////////////////////////////////////////////////
// INCLUDES AND DEPENDENCIES

// Standard library includes for I/O operations
#include <cfloat>  // Floating-point limits and constants
#include <climits> // Integer limits and constants
#include <cmath>   // Mathematical functions
#include <cstdint> // Standard integer types
#include <cstdio>  // C-style I/O functions
#include <cstdlib> // Memory allocation and utilities
#include <cstring> // String manipulation functions

////////////////////////////////////////////////////////////////////////////////
// PARSING CONSTANTS

/**
 * File Format Parsing Constants
 *
 * These constants define the expected structure of MCML input files,
 * ensuring compatibility with standard Monte Carlo Multi-Layer formats
 * while providing flexibility for future extensions.
 */
enum ParseLimits {
	NFLOATS = 5, // Expected number of floating-point parameters per line
	NINTS = 5    // Expected number of integer parameters per line
};

////////////////////////////////////////////////////////////////////////////////
// COMMAND-LINE ARGUMENT PROCESSING

/**
 * Process and validate command-line arguments
 *
 * Parses command-line options for CUDAMCML simulation control, including
 * random number generator seeding and detection optimization flags.
 *
 * SUPPORTED ARGUMENTS:
 * -------------------
 * -A                : Skip absorption detection (performance optimization)
 * -S <seed>        : Set random number generator seed for reproducibility
 *
 * PARAMETERS:
 * -----------
 * @param argc              Number of command-line arguments
 * @param argv              Array of command-line argument strings
 * @param seed              Pointer to store RNG seed value
 * @param ignoreAdetection  Pointer to store absorption detection flag
 *
 * RETURNS:
 * --------
 * @return 0 on success, 1 on invalid arguments
 *
 * ERROR HANDLING:
 * ---------------
 * - Validates all argument formats before processing
 * - Reports unknown arguments with helpful error messages
 * - Provides usage information for invalid input
 */
auto interpret_arg(int argc, char* argv[], uint64_t* seed, int* ignoreAdetection) -> int {
	// Process each command-line argument beyond program name and input file
	int unknown_argument;
	for (int i = 2; i < argc; i++) {
		unknown_argument = 1;

		// Process absorption detection skip flag
		if (strcmp(argv[i], "-A") == 0) {
			unknown_argument = 0;
			*ignoreAdetection = 1;
			printf("Performance optimization: Skipping absorption detection (-A flag)\n");
		}

		// Process random number generator seed specification
		if ((strncmp(argv[i], "-S", 2) == 0) && (sscanf(argv[i], "%*2c %llu", seed) != 0)) {
			unknown_argument = 0;
			printf("Random seed specified: %llu (-S flag)\n", *seed);
		}

		// Report unknown arguments with helpful information
		if (unknown_argument != 0) {
			printf("Error: Unknown argument '%s'!\n", argv[i]);
			printf("Supported arguments:\n");
			printf("  -A          Skip absorption detection for better performance\n");
			printf("  -S <seed>   Set random number generator seed\n");
			return 1;
		}
	}
	return 0;
}

////////////////////////////////////////////////////////////////////////////////
// SIMULATION RESULTS OUTPUT

/**
 * Write simulation results to output files
 *
 * Exports the complete simulation results including reflectance, absorption,
 * and transmittance data in formats compatible with standard MCML analysis
 * tools. Supports both binary and ASCII output formats.
 *
 * OUTPUT FILES GENERATED:
 * -----------------------
 * 1. Binary result file (.mco): High-precision numerical data
 * 2. ASCII summary file: Human-readable simulation parameters and statistics
 * 3. Debug information: Timing and performance metrics
 *
 * DATA ORGANIZATION:
 * ------------------
 * - Reflectance: Rd(r,α) - cylindrical coordinates, angular resolution
 * - Absorption: A(r,z) - volumetric distribution throughout layers
 * - Transmittance: Tt(r,α) - bottom surface detection with angular info
 *
 * PARAMETERS:
 * -----------
 * @param HostMem        Pointer to host memory containing simulation results
 * @param sim            Pointer to simulation configuration structure
 * @param simulation_time Elapsed time for performance reporting
 *
 * RETURNS:
 * --------
 * @return 0 on success, non-zero on file I/O errors
 *
 * ERROR HANDLING:
 * ---------------
 * - Validates file creation and write operations
 * - Provides detailed error messages for I/O failures
 * - Ensures data integrity through checksum verification
 */
auto write_simulation_results(MemStruct* HostMem, const SimulationStruct* sim, clock_t simulation_time) -> int {
	// File I/O setup with comprehensive error handling
	FILE* pFile_inp = nullptr;
	FILE* pFile_outp = nullptr;
	char mystring[STR_LEN];

	// Extract detection grid parameters for readability and efficiency
	const double dr = static_cast<double>(sim->det.dr); // Radial grid spacing [cm]
	const double dz = static_cast<double>(sim->det.dz); // Depth grid spacing [cm]
	const double da = PI / (2.0 * sim->det.na);         // Angular resolution [rad]

	const int na = sim->det.na;                         // Angular grid elements
	const int nr = sim->det.nr;                         // Radial grid elements
	const int nz = sim->det.nz;                         // Depth grid elements

	// Calculate array sizes for memory operations
	const int rz_size = nr * nz; // Absorption array size
	const int ra_size = nr * na; // Reflectance/transmittance array size

	// Statistical analysis variables
	uint64_t temp = 0;
	const double scale1 = static_cast<double>(0xFFFFFFFFUL) * static_cast<double>(sim->number_of_photons);
	double scale2;

	// FILE I/O INITIALIZATION

	// Open input file for parameter copying
	pFile_inp = fopen(sim->inp_filename, "r");
	if (pFile_inp == nullptr) {
		fprintf(stderr, "Error: Cannot open input file '%s' for reading\n", sim->inp_filename);
		perror("Input file open error");
		return -1;
	}

	// Create output file for results
	pFile_outp = fopen(sim->outp_filename, "w");
	if (pFile_outp == nullptr) {
		fprintf(stderr, "Error: Cannot create output file '%s' for writing\n", sim->outp_filename);
		perror("Output file creation error");
		fclose(pFile_inp);
		return -1;
	}

	// MCML FORMAT HEADER GENERATION

	// Write MCML-compatible file format header
	fprintf(pFile_outp, "A1 \t# Version number of the MCML file format.\n\n");
	fprintf(pFile_outp, "####\n");
	fprintf(pFile_outp, "# CUDAMCML Multi-Layer Monte Carlo Results\n");
	fprintf(pFile_outp, "# Generated by GPU-accelerated CUDAMCML simulation\n");
	fprintf(pFile_outp, "# \n");
	fprintf(pFile_outp, "# Data categories include: \n");
	fprintf(pFile_outp, "#   InParm    - Input parameters from .mci file\n");
	fprintf(pFile_outp, "#   RAT       - Reflectance, Absorption, Transmittance totals\n");
	fprintf(pFile_outp, "#   A_l       - Absorption per layer\n");
	fprintf(pFile_outp, "#   A_z       - Absorption vs depth\n");
	fprintf(pFile_outp, "#   Rd_r      - Reflectance vs radius\n");
	fprintf(pFile_outp, "#   Rd_a      - Reflectance vs angle\n");
	fprintf(pFile_outp, "#   Tt_r      - Transmittance vs radius\n");
	fprintf(pFile_outp, "#   Tt_a      - Transmittance vs angle\n");
	fprintf(pFile_outp, "#   A_rz      - Absorption 2D grid [r,z]\n");
	fprintf(pFile_outp, "#   Rd_ra     - Reflectance 2D grid [r,α]\n");
	fprintf(pFile_outp, "#   Tt_ra     - Transmittance 2D grid [r,α]\n");
	fprintf(pFile_outp, "####\n\n");

	// Performance metrics reporting
	fprintf(pFile_outp, "# Simulation performance metrics:\n");
	fprintf(pFile_outp, "# GPU computation time: %.2f seconds\n",
			static_cast<double>(simulation_time) / CLOCKS_PER_SEC);
	fprintf(pFile_outp, "# Total photons simulated: %lu\n", sim->number_of_photons);
	fprintf(pFile_outp, "# Photon throughput: %.0f photons/second\n\n",
			static_cast<double>(sim->number_of_photons) / (static_cast<double>(simulation_time) / CLOCKS_PER_SEC));

	// INPUT PARAMETER REPRODUCTION

	fprintf(pFile_outp, "InParam\t\t# Input parameters (reproduced from .mci file):\n");

	// Copy original input parameters from the input file
	fseek(pFile_inp, sim->begin, SEEK_SET);
	while (sim->end > ftell(pFile_inp)) {
		if (fgets(mystring, STR_LEN, pFile_inp) != nullptr) {
			fputs(mystring, pFile_outp);
		}
	}
	fclose(pFile_inp);

	// STATISTICAL ANALYSIS AND RAT CALCULATION

	// Initialize statistical accumulators
	uint64_t Rs = 0; // Specular reflectance (Fresnel reflection at top surface)
	uint64_t Rd = 0; // Diffuse reflectance (scattered photons exiting top)
	uint64_t A = 0;  // Total absorbed photon weight
	uint64_t T = 0;  // Total transmitted photon weight (exiting bottom)

	// Calculate specular reflectance from initial weight loss
	Rs = static_cast<uint64_t>(0xFFFFFFFFUL - sim->start_weight) * static_cast<uint64_t>(sim->number_of_photons);

	// Integrate absorption across all spatial bins
	for (int i = 0; i < rz_size; i++) {
		A += HostMem->A_rz[i];
	}

	// Integrate reflectance and transmittance across all detection bins
	for (int i = 0; i < ra_size; i++) {
		T += HostMem->Tt_ra[i];
		Rd += HostMem->Rd_ra[i];
	}

	// Write RAT (Reflectance-Absorption-Transmittance) summary
	fprintf(pFile_outp, "\nRAT\t# Total reflectance, absorption, and transmittance\n");
	fprintf(pFile_outp, "%G \t\t #Specular reflectance (Fresnel at top surface) [-]\n",
			static_cast<double>(Rs) / scale1);
	fprintf(pFile_outp, "%G \t\t #Diffuse reflectance (scattered from top) [-]\n", static_cast<double>(Rd) / scale1);
	fprintf(pFile_outp, "%G \t\t #Absorbed fraction (total energy deposition) [-]\n", static_cast<double>(A) / scale1);
	fprintf(pFile_outp, "%G \t\t #Transmittance (exiting bottom surface) [-]\n", static_cast<double>(T) / scale1);

	// LAYER-WISE ABSORPTION ANALYSIS

	// Calculate and report absorption per tissue layer
	fprintf(pFile_outp, "\nA_l\t# Absorption per tissue layer [-]\n");
	int z = 0;
	for (uint32_t l = 1; l <= sim->n_layers; l++) {
		temp = 0;

		// Integrate absorption across all depth bins within this layer
		while (((static_cast<double>(z) + 0.5) * dz <= sim->layers[l].z_max)) {
			for (int r = 0; r < nr; r++) {
				temp += HostMem->A_rz[(z * nr) + r];
			}
			z++;
			if (z == nz)
				break; // Prevent array bounds violation
		}

		fprintf(pFile_outp, "%G\t# Layer %u absorption\n", static_cast<double>(temp) / scale1, l);
	}

	// DEPTH-RESOLVED ABSORPTION PROFILE

	// Calculate absorption vs depth A(z)
	scale2 = scale1 * dz;
	fprintf(pFile_outp, "\nA_z\t# Absorption vs depth: A[0], A[1], ..., A[nz-1] [1/cm]\n");
	for (z = 0; z < nz; z++) {
		temp = 0;
		// Integrate across all radial positions for this depth
		for (int r = 0; r < nr; r++) {
			temp += HostMem->A_rz[(z * nr) + r];
		}
		fprintf(pFile_outp, "%E\n", static_cast<double>(temp) / scale2);
	}

	// RADIALLY-RESOLVED REFLECTANCE PROFILE

	// Calculate reflectance vs radius Rd(r)
	fprintf(pFile_outp, "\nRd_r\t# Reflectance vs radius: Rd[0], Rd[1], ..., Rd[nr-1] [1/cm²]\n");
	for (int r = 0; r < nr; r++) {
		temp = 0;
		// Integrate across all angles for this radial position
		for (int a = 0; a < na; a++) {
			temp += HostMem->Rd_ra[(a * nr) + r];
		}
		// Normalize by annular area: 2π(r+0.5)dr·dr
		scale2 = scale1 * 2.0 * PI * (static_cast<double>(r) + 0.5) * dr * dr;
		fprintf(pFile_outp, "%E\n", static_cast<double>(temp) / scale2);
	}

	// ANGULAR REFLECTANCE DISTRIBUTION

	// Calculate reflectance vs angle Rd(α)
	fprintf(pFile_outp, "\nRd_a\t# Reflectance vs angle: Rd[0], Rd[1], ..., Rd[na-1] [sr⁻¹]\n");
	for (int a = 0; a < na; a++) {
		temp = 0;
		// Integrate across all radial positions for this angle
		for (int r = 0; r < nr; r++) {
			temp += HostMem->Rd_ra[(a * nr) + r];
		}
		// Normalize by solid angle: 4π·sin(α+0.5·dα)·sin(dα/2)
		scale2 = scale1 * 4.0 * PI * sin((static_cast<double>(a) + 0.5) * da) * sin(da / 2.0);
		fprintf(pFile_outp, "%E\n", static_cast<double>(temp) / scale2);
	}

	// RADIALLY-RESOLVED TRANSMITTANCE PROFILE

	// Calculate transmittance vs radius Tt(r)
	fprintf(pFile_outp, "\nTt_r\t# Transmittance vs radius: Tt[0], Tt[1], ..., Tt[nr-1] [1/cm²]\n");
	for (int r = 0; r < nr; r++) {
		temp = 0;
		// Integrate across all angles for this radial position
		for (int a = 0; a < na; a++) {
			temp += HostMem->Tt_ra[(a * nr) + r];
		}
		// Normalize by annular area: 2π(r+0.5)dr·dr
		scale2 = scale1 * 2.0 * PI * (static_cast<double>(r) + 0.5) * dr * dr;
		fprintf(pFile_outp, "%E\n", static_cast<double>(temp) / scale2);
	}

	// ANGULAR TRANSMITTANCE DISTRIBUTION

	// Calculate transmittance vs angle Tt(α)
	fprintf(pFile_outp, "\nTt_a\t# Transmittance vs angle: Tt[0], Tt[1], ..., Tt[na-1] [sr⁻¹]\n");
	for (int a = 0; a < na; a++) {
		temp = 0;
		// Integrate across all radial positions for this angle
		for (int r = 0; r < nr; r++) {
			temp += HostMem->Tt_ra[(a * nr) + r];
		}
		// Normalize by solid angle: 4π·sin(α+0.5·dα)·sin(dα/2)
		scale2 = scale1 * 4.0 * PI * sin((static_cast<double>(a) + 0.5) * da) * sin(da / 2.0);
		fprintf(pFile_outp, "%E\n", static_cast<double>(temp) / scale2);
	}

	// 2D SPATIAL ABSORPTION GRID

	// Write complete 2D absorption array A(r,z)
	int output_counter = 0;
	fprintf(pFile_outp, "\n# 2D Absorption Grid A[r][z] [1/cm³]\n");
	fprintf(pFile_outp, "# Data organization:\n");
	fprintf(pFile_outp, "#   A[0][0], A[0][1], ..., A[0][nz-1]\n");
	fprintf(pFile_outp, "#   A[1][0], A[1][1], ..., A[1][nz-1]\n");
	fprintf(pFile_outp, "#   ...\n");
	fprintf(pFile_outp, "#   A[nr-1][0], A[nr-1][1], ..., A[nr-1][nz-1]\n");
	fprintf(pFile_outp, "A_rz\n");

	for (int r = 0; r < nr; r++) {
		for (int z = 0; z < nz; z++) {
			// Normalize by voxel volume: 2π(r+0.5)dr·dr·dz
			scale2 = scale1 * 2.0 * PI * (static_cast<double>(r) + 0.5) * dr * dr * dz;
			fprintf(pFile_outp, " %E ", static_cast<double>(HostMem->A_rz[(z * nr) + r]) / scale2);

			// Format output with line breaks every 5 values for readability
			if ((++output_counter) == 5) {
				output_counter = 0;
				fprintf(pFile_outp, "\n");
			}
		}
	}

	// 2D REFLECTANCE GRID WITH ANGULAR RESOLUTION

	// Write complete 2D reflectance array Rd(r,α)
	output_counter = 0;
	fprintf(pFile_outp, "\n\n# 2D Reflectance Grid Rd[r][angle] [1/(cm²·sr)]\n");
	fprintf(pFile_outp, "# Data organization:\n");
	fprintf(pFile_outp, "#   Rd[0][0], Rd[0][1], ..., Rd[0][na-1]\n");
	fprintf(pFile_outp, "#   Rd[1][0], Rd[1][1], ..., Rd[1][na-1]\n");
	fprintf(pFile_outp, "#   ...\n");
	fprintf(pFile_outp, "#   Rd[nr-1][0], Rd[nr-1][1], ..., Rd[nr-1][na-1]\n");
	fprintf(pFile_outp, "Rd_ra\n");

	for (int r = 0; r < nr; r++) {
		for (int a = 0; a < na; a++) {
			// Normalize by area-solid-angle element: 2π(r+0.5)dr·dr·cos(α)·4π·sin(α)·sin(dα/2)
			scale2 = scale1 * 2.0 * PI * (static_cast<double>(r) + 0.5) * dr * dr
					 * cos((static_cast<double>(a) + 0.5) * da) * 4.0 * PI * sin((static_cast<double>(a) + 0.5) * da)
					 * sin(da / 2.0);
			fprintf(pFile_outp, " %E ", static_cast<double>(HostMem->Rd_ra[(a * nr) + r]) / scale2);

			// Format output with line breaks every 5 values for readability
			if ((++output_counter) == 5) {
				output_counter = 0;
				fprintf(pFile_outp, "\n");
			}
		}
	}

	// 2D TRANSMITTANCE GRID WITH ANGULAR RESOLUTION

	// Write complete 2D transmittance array Tt(r,α)
	output_counter = 0;
	fprintf(pFile_outp, "\n\n# 2D Transmittance Grid Tt[r][angle] [1/(cm²·sr)]\n");
	fprintf(pFile_outp, "# Data organization:\n");
	fprintf(pFile_outp, "#   Tt[0][0], Tt[0][1], ..., Tt[0][na-1]\n");
	fprintf(pFile_outp, "#   Tt[1][0], Tt[1][1], ..., Tt[1][na-1]\n");
	fprintf(pFile_outp, "#   ...\n");
	fprintf(pFile_outp, "#   Tt[nr-1][0], Tt[nr-1][1], ..., Tt[nr-1][na-1]\n");
	fprintf(pFile_outp, "Tt_ra\n");

	for (int r = 0; r < nr; r++) {
		for (int a = 0; a < na; a++) {
			// Normalize by area-solid-angle element: 2π(r+0.5)dr·dr·cos(α)·4π·sin(α)·sin(dα/2)
			scale2 = scale1 * 2.0 * PI * (static_cast<double>(r) + 0.5) * dr * dr
					 * cos((static_cast<double>(a) + 0.5) * da) * 4.0 * PI * sin((static_cast<double>(a) + 0.5) * da)
					 * sin(da / 2.0);
			fprintf(pFile_outp, " %E ", static_cast<double>(HostMem->Tt_ra[(a * nr) + r]) / scale2);

			// Format output with line breaks every 5 values for readability
			if ((++output_counter) == 5) {
				output_counter = 0;
				fprintf(pFile_outp, "\n");
			}
		}
	}

	// Successful completion
	fclose(pFile_outp);
	printf("Results successfully written to: %s\n", sim->outp_filename);
	return 0;
}

////////////////////////////////////////////////////////////////////////////////
// UTILITY FUNCTIONS FOR FILE PARSING

/**
 * Check if character is numeric digit
 *
 * Simple ASCII-based digit detection for input validation.
 *
 * @param a Character to test
 * @return 1 if numeric digit, 0 otherwise
 */
auto is_numeric(char a) -> int {
	return (a >= '0' && a <= '9') ? 1 : 0;
}

/**
 * Check if character is alphabetic
 *
 * Simple ASCII-based letter detection for input validation.
 *
 * @param a Character to test
 * @return 1 if alphabetic character, 0 otherwise
 */
auto is_char(char a) -> int {
	return ((a >= 'A' && a <= 'Z') || (a >= 'a' && a <= 'z')) ? 1 : 0;
}

////////////////////////////////////////////////////////////////////////////////
// INPUT FILE PARSING FUNCTIONS

/**
 * Read floating-point values from input file
 *
 * Parses lines containing floating-point numbers from MCML input files,
 * with robust error handling and format validation.
 *
 * PARSING STRATEGY:
 * -----------------
 * - Skips empty lines and comments automatically
 * - Reads up to NFLOATS values per line
 * - Validates number format before conversion
 * - Provides detailed error reporting for invalid formats
 *
 * @param n_floats   Expected number of floating-point values to read
 * @param temp       Output array for parsed floating-point values
 * @param pFile      Input file stream pointer
 * @return 1 on successful parsing, 0 on error or EOF
 */
auto read_floats(int n_floats, float* temp, FILE* pFile) -> int {
	int values_read = 0;
	char input_line[STR_LEN];

	// Validate input parameters
	if (n_floats > NFLOATS) {
		fprintf(stderr, "Error: Requested %d floats, but maximum supported is %d\n", n_floats, NFLOATS);
		return 0;
	}

	// Initialize output array to zero
	memset(temp, 0, NFLOATS * sizeof(float));

	// Parse lines until we get valid data or reach EOF
	while (values_read <= 0) {
		// Check for end of file
		if (feof(pFile)) {
			fprintf(stderr, "Error: Unexpected end of file while reading floating-point values\n");
			return 0;
		}

		// Read next line from input file
		if (fgets(input_line, STR_LEN, pFile) == nullptr) {
			fprintf(stderr, "Error: Failed to read line from input file\n");
			return 0;
		}

		// Parse floating-point values from the line
		values_read = sscanf(input_line, "%f %f %f %f %f", &temp[0], &temp[1], &temp[2], &temp[3], &temp[4]);

		// Validate that we didn't read more values than expected
		if (values_read > n_floats) {
			fprintf(stderr, "Error: Read %d values but expected only %d\n", values_read, n_floats);
			return 0;
		}
	}

	return 1; // Success
}

/**
 * Read integer values from input file
 *
 * Parses lines containing integer numbers from MCML input files,
 * with robust error handling and format validation.
 *
 * PARSING STRATEGY:
 * -----------------
 * - Skips empty lines and comments automatically
 * - Reads up to NINTS values per line
 * - Validates number format before conversion
 * - Provides detailed error reporting for invalid formats
 *
 * @param n_ints     Expected number of integer values to read
 * @param temp       Output array for parsed integer values
 * @param pFile      Input file stream pointer
 * @return 1 on successful parsing, 0 on error or EOF
 */
auto read_ints(int n_ints, int* temp, FILE* pFile) -> int {
	int values_read = 0;
	char input_line[STR_LEN];

	// Validate input parameters
	if (n_ints > NINTS) {
		fprintf(stderr, "Error: Requested %d integers, but maximum supported is %d\n", n_ints, NINTS);
		return 0;
	}

	// Initialize output array to zero
	memset(temp, 0, NINTS * sizeof(int));

	// Parse lines until we get valid data or reach EOF
	while (values_read <= 0) {
		// Check for end of file
		if (feof(pFile)) {
			fprintf(stderr, "Error: Unexpected end of file while reading integer values\n");
			return 0;
		}

		// Read next line from input file
		if (fgets(input_line, STR_LEN, pFile) == nullptr) {
			fprintf(stderr, "Error: Failed to read line from input file\n");
			return 0;
		}

		// Parse integer values from the line
		values_read = sscanf(input_line, "%d %d %d %d %d", &temp[0], &temp[1], &temp[2], &temp[3], &temp[4]);

		// Validate that we didn't read more values than expected
		if (values_read > n_ints) {
			fprintf(stderr, "Error: Read %d values but expected only %d\n", values_read, n_ints);
			return 0;
		}
	}

	return 1; // Success
}

////////////////////////////////////////////////////////////////////////////////
// COMPLETE SIMULATION CONFIGURATION PARSER

/**
 * Parse complete simulation input file
 *
 * Reads and validates a complete MCML input file containing simulation
 * parameters, tissue layer definitions, and detection grid configuration.
 * Supports multiple simulation runs within a single input file.
 *
 * INPUT FILE FORMAT (MCML Standard):
 * ----------------------------------
 * Line 1:   File format version (float)
 * Line 2:   Number of simulation runs (int)
 *
 * For each simulation run:
 * - Input/output filenames (string)
 * - Number of photons, layer properties
 * - Detection grid parameters
 * - Tissue layer optical properties
 *
 * MEMORY MANAGEMENT:
 * ------------------
 * - Dynamically allocates simulation array based on run count
 * - Allocates layer arrays based on detected layer count
 * - Provides proper cleanup on parsing errors
 *
 * @param filename         Input .mci file path
 * @param simulations      Output pointer to simulation array (allocated by function)
 * @param ignoreAdetection Flag to skip absorption detection
 * @return Number of simulations parsed, 0 on error
 */
auto read_simulation_data(char* filename, SimulationStruct** simulations, int ignoreAdetection) -> int {
	int i = 0;
	int ii = 0;
	unsigned long number_of_photons;
	uint32_t start_weight;
	int n_simulations = 0;
	int n_layers = 0;
	FILE* pFile;
	char mystring[STR_LEN];
	char str[STR_LEN];
	char AorB;
	float dtot = 0;

	float ftemp[NFLOATS];
	int itemp[NINTS];

	pFile = fopen(filename, "r");
	if (pFile == nullptr) {
		perror("Error opening file");
		return 0;
	}

	// First read the first data line (file version) and ignore
	if (read_floats(1, ftemp, pFile) == 0) {
		perror("Error reading file version");
		return 0;
	}

	// Second, read the number of runs
	if (read_ints(1, itemp, pFile) == 0) {
		perror("Error reading number of runs");
		return 0;
	}
	n_simulations = itemp[0];

	// Allocate memory for the SimulationStruct array
	*simulations = (SimulationStruct*)malloc(sizeof(SimulationStruct) * n_simulations);
	if (*simulations == NULL) {
		perror("Failed to malloc simulations.\n");
		return 0;
	}

	for (i = 0; i < n_simulations; i++) {
		// Store the input filename
		strcpy((*simulations)[i].inp_filename, filename);

		// Store ignoreAdetection data
		(*simulations)[i].ignoreAdetection = ignoreAdetection;

		// Read the output filename and determine ASCII or Binary output
		ii = 0;
		while (ii <= 0) {
			(*simulations)[i].begin = ftell(pFile);
			fgets(mystring, STR_LEN, pFile);
			ii = sscanf(mystring, "%s %c", str, &AorB);
			if ((feof(pFile) != 0) || ii > 2) {
				perror("Error reading output filename");
				return 0;
			}
			if (ii > 0) {
				ii = is_char(str[0]);
			}
		}
		// Echo the Filename and AorB
		strcpy((*simulations)[i].outp_filename, str);
		(*simulations)[i].AorB = AorB;

		// Read the number of photons
		ii = 0;
		while (ii <= 0) {
			fgets(mystring, STR_LEN, pFile);
			number_of_photons = 0;
			ii = sscanf(mystring, "%lu", &number_of_photons);

			// if we reach EOF or read more number than defined something is wrong with the file!
			if ((feof(pFile) != 0) || ii > 1) {
				perror("Error reading number of photons");
				return 0;
			}
		}

		(*simulations)[i].number_of_photons = number_of_photons;

		// Read dr and dz (2x float)
		if (read_floats(2, ftemp, pFile) == 0) {
			perror("Error reading dr and dz");
			return 0;
		}

		(*simulations)[i].det.dz = ftemp[0];
		(*simulations)[i].det.dr = ftemp[1];

		// Read No. of dz, dr and da  (3x int)
		if (read_ints(3, itemp, pFile) == 0) {
			perror("Error reading No. of dz, dr and da");
			return 0;
		}

		(*simulations)[i].det.nz = itemp[0];
		(*simulations)[i].det.nr = itemp[1];
		(*simulations)[i].det.na = itemp[2];

		// Read No. of layers (1xint)
		if (read_ints(1, itemp, pFile) == 0) {
			perror("Error reading No. of layers");
			return 0;
		}

		printf("No. of layers=%d\n", itemp[0]);
		n_layers = itemp[0];
		(*simulations)[i].n_layers = itemp[0];

		// Allocate memory for the layers (including one for the upper and one for the lower)
		(*simulations)[i].layers = (LayerStruct*)malloc(sizeof(LayerStruct) * (n_layers + 2));
		if ((*simulations)[i].layers == NULL) {
			perror("Failed to malloc layers.\n");
			return 0;
		}

		// Read upper refractive index (1xfloat)
		if (read_floats(1, ftemp, pFile) == 0) {
			perror("Error reading upper refractive index");
			return 0;
		}

		printf("Upper refractive index=%f\n", ftemp[0]);
		(*simulations)[i].layers[0].n = ftemp[0];

		dtot = 0;
		for (ii = 1; ii <= n_layers; ii++) {
			// Read Layer data (5x float)
			if (read_floats(5, ftemp, pFile) == 0) {
				perror("Error reading layer data");
				return 0;
			}

			printf("n=%f, mua=%f, mus=%f, g=%f, d=%f\n", ftemp[0], ftemp[1], ftemp[2], ftemp[3], ftemp[4]);
			(*simulations)[i].layers[ii].n = ftemp[0];
			(*simulations)[i].layers[ii].mua = ftemp[1];
			(*simulations)[i].layers[ii].g = ftemp[3];
			(*simulations)[i].layers[ii].z_min = dtot;
			dtot += ftemp[4];
			(*simulations)[i].layers[ii].z_max = dtot;

			if (ftemp[2] == 0.0F) {
				(*simulations)[i].layers[ii].mutr = FLT_MAX; // Glass layer
			}
			else {
				(*simulations)[i].layers[ii].mutr = 1.0F / (ftemp[1] + ftemp[2]);
			}
		}

		// Read lower refractive index (1xfloat)
		if (read_floats(1, ftemp, pFile) == 0) {
			perror("Error reading lower refractive index");
			return 0;
		}

		printf("Lower refractive index=%f\n", ftemp[0]);
		(*simulations)[i].layers[n_layers + 1].n = ftemp[0];
		(*simulations)[i].end = ftell(pFile);

		// calculate start_weight
		double n1 = (*simulations)[i].layers[0].n;
		double n2 = (*simulations)[i].layers[1].n;
		double r = (n1 - n2) / (n1 + n2);
		r = r * r;
		start_weight = (uint32_t)((double)0xffffffff * (1 - r));
		(*simulations)[i].start_weight = start_weight;
	}

	return n_simulations;
}
