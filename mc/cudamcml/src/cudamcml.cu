#include "hip/hip_runtime.h"
/*==============================================================================
 * CUDAMCML - GPU-Accelerated Monte Carlo Multi-Layer Photon Transport
 *
 * CUDA-based Monte Carlo simulation of photon migration in layered media (CUDAMCML).
 *
 * This implementation extends Monte Carlo photon transport to complex multi-layered
 * biological tissues with arbitrary optical properties. The GPU acceleration enables
 * high-resolution simulations that would be computationally prohibitive on CPU.
 *
 * HISTORICAL CONTEXT:
 * -------------------
 * Some documentation is available for CUDAMCML and should have been distributed along
 * with this source code. If that is not the case: Documentation, source code and
 * executables for CUDAMCML are available for download on our webpage:
 *
 * http://www.atomic.physics.lu.se/Biophotonics
 * http://www.atomic.physics.lu.se/fileadmin/atomfysik/Biophotonics/Software/CUDAMCML.zip
 *
 * We encourage the use, and modification of this code, and hope it will help
 * users/programmers to utilize the power of GPGPU for their simulation needs. While we
 * don't have a scientific publication describing this code, we would very much appreciate
 * if you cite our original GPGPU Monte Carlo letter (on which CUDAMCML is based) if you
 * use this code or derivations thereof for your own scientific work:
 *
 * E. Alerstam, T. Svensson and S. Andersson-Engels, "Parallel computing with graphics
 * processing units for high-speed Monte Carlo simulations of photon migration", Journal
 * of Biomedical Optics Letters, 13(6) 060504 (2008).
 *
 * PHYSICAL MODEL:
 * ---------------
 * - Multi-layered turbid media with configurable layer properties
 * - Pencil beam or collimated source incident on top surface
 * - Henyey-Greenberg scattering with layer-specific anisotropy factors
 * - Fresnel reflection/transmission at all refractive index boundaries
 * - Spatially-resolved detection: Rd(r,α), A(r,z), Tt(r,α)
 * - Russian roulette photon termination for computational efficiency
 *
 * COMPUTATIONAL ARCHITECTURE:
 * ---------------------------
 * - Modular design: Separate compilation units for I/O, memory, RNG, transport
 * - Dynamic GPU configuration: Runtime adaptation to hardware capabilities
 * - Memory optimization: Aligned structures, constant memory utilization
 * - Exception-safe programming: RAII principles with modern C++
 * - Comprehensive error handling: CUDA-specific diagnostics and recovery
 *
 * SIMULATION WORKFLOW:
 * --------------------
 * 1. Parse input file with layer properties and simulation parameters
 * 2. Configure GPU based on detected hardware capabilities
 * 3. Allocate and initialize GPU memory structures
 * 4. Launch massively parallel photon transport kernels
 * 5. Collect detection results and perform statistical analysis
 * 6. Export results in standard MCML format
 *
 * LICENSE:
 * --------
 * This file is part of CUDAMCML.
 *
 * CUDAMCML is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * CUDAMCML is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with CUDAMCML.  If not, see <http://www.gnu.org/licenses/>.
 *
 * This code is distributed under the terms of the GNU General Public Licence.
 */

////////////////////////////////////////////////////////////////////////////////
// SYSTEM INCLUDES AND DEPENDENCIES

// Standard C++ library includes for I/O, memory management, and error handling
#include <cfloat>    // Floating-point limits and constants
#include <cstdio>    // C-style I/O for compatibility
#include <iomanip>   // Stream manipulators for formatted output
#include <iostream>  // Modern C++ I/O streams
#include <memory>    // Smart pointers and memory management
#include <stdexcept> // Standard exception classes

// CUDA runtime and device includes for GPU computing
#include <hip/hip_runtime.h>             // CUDA runtime API
#include <> // CUDA kernel launch parameters

// Project-specific includes
#include "cudamcml.h"    // Main header with constants, structures, and function declarations
#include "safe_primes.h" // Embedded safe prime numbers for RNG initialization

// Include specialized component modules - compiled as single translation unit
#include "cudamcml_io.cu"        // Input/output operations and file parsing
#include "cudamcml_mem.cu"       // GPU memory management and allocation
#include "cudamcml_rng.cu"       // Random number generation and initialization
#include "cudamcml_transport.cu" // Core photon transport physics and detection

////////////////////////////////////////////////////////////////////////////////
// CUDA MONTE CARLO MULTI-LAYER NAMESPACE

namespace cudamcml
{

////////////////////////////////////////////////////////////////////////////////
// GPU CONFIGURATION AND HARDWARE MANAGEMENT

/**
 * GPU Configuration Management Class
 *
 * Encapsulates CUDA hardware detection, configuration, and optimization
 * for multi-layered Monte Carlo photon transport simulations. Adapts
 * thread configuration based on detected GPU capabilities.
 *
 * DESIGN PHILOSOPHY:
 * - Dynamic configuration based on actual hardware capabilities
 * - Conservative defaults for broad GPU compatibility
 * - Oversubscription for optimal occupancy and throughput
 * - Detailed hardware reporting for performance analysis
 *
 * CONFIGURATION STRATEGY:
 * - Detect multiprocessor count and capabilities
 * - Apply oversubscription factor for improved occupancy
 * - Balance thread count with memory requirements
 * - Validate against hardware limits and constraints
 */
class GPUConfig
{
public:
	// Conservative fallback configuration for unknown hardware
	static constexpr int DEFAULT_FALLBACK_BLOCKS = 56;
	static constexpr int DEFAULT_FALLBACK_THREADS = 28672;

	// Active GPU configuration parameters
	int num_blocks;        // Number of CUDA thread blocks
	int threads_per_block; // Threads per block (typically 512)
	int total_threads;     // Total concurrent threads

	/**
	 * Constructor - Initialize with fallback configuration
	 *
	 * Uses conservative defaults that work across different GPU generations.
	 * Actual configuration is determined during initialize() call.
	 */
	GPUConfig() :
		num_blocks(DEFAULT_FALLBACK_BLOCKS), threads_per_block(DEFAULT_THREADS_PER_BLOCK),
		total_threads(DEFAULT_FALLBACK_THREADS) {}

	/**
	 * Hardware Detection and Dynamic Configuration
	 *
	 * Detects GPU capabilities and configures optimal thread parameters
	 * for maximum throughput. Applies oversubscription and validates
	 * against hardware constraints.
	 */
	void initialize() {
		hipDeviceProp_t deviceProp;
		int device;

		CUDA_CHECK_ERROR(hipGetDevice(&device));
		CUDA_CHECK_ERROR(hipGetDeviceProperties(&deviceProp, device));

		// Extract key hardware characteristics
		const int multiprocessors = deviceProp.multiProcessorCount;
		const int max_threads_per_sm = deviceProp.maxThreadsPerMultiProcessor;
		const int max_blocks_per_sm = deviceProp.maxBlocksPerMultiProcessor;

		// Calculate optimal thread configuration for multi-layered Monte Carlo
		threads_per_block = DEFAULT_THREADS_PER_BLOCK;

		// Apply oversubscription for improved GPU utilization
		// More blocks than SMs helps hide memory latency and improve throughput
		num_blocks = multiprocessors * GPU_OVERSUBSCRIPTION_FACTOR;
		total_threads = num_blocks * threads_per_block;

		print_configuration(deviceProp, multiprocessors, max_threads_per_sm, max_blocks_per_sm);
		copy_to_device_constants();
	}

private:
	/**
	 * Print comprehensive GPU configuration report
	 *
	 * Displays detected hardware capabilities, configured parameters,
	 * and oversubscription strategy for performance optimization.
	 */
	void print_configuration(const hipDeviceProp_t& deviceProp, int multiprocessors, int max_threads_per_sm,
							 int max_blocks_per_sm) const {
		std::cout << "=== CUDAMCML GPU Configuration ===\n"
				  << "GPU: " << deviceProp.name << "\n"
				  << "Compute Capability: " << deviceProp.major << "." << deviceProp.minor << "\n"
				  << "Multiprocessors: " << multiprocessors << "\n"
				  << "Max threads per SM: " << max_threads_per_sm << "\n"
				  << "Max blocks per SM: " << max_blocks_per_sm << "\n"
				  << "Configured blocks: " << num_blocks << "\n"
				  << "Threads per block: " << threads_per_block << "\n"
				  << "Total GPU threads: " << total_threads << "\n"
				  << "Oversubscription factor: " << GPU_OVERSUBSCRIPTION_FACTOR << "x\n"
				  << "Memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB\n"
				  << "===================================\n\n";
	}

	/**
	 * Copy configuration to GPU constant memory
	 *
	 * Transfers thread configuration parameters to constant memory
	 * for high-performance access from device kernels.
	 */
	void copy_to_device_constants() const {
		CUDA_CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_threads_per_block), &threads_per_block, sizeof(int)));
		CUDA_CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_total_threads), &total_threads, sizeof(int)));
	}
};

} // namespace cudamcml

////////////////////////////////////////////////////////////////////////////////
// GLOBAL STATE MANAGEMENT AND LEGACY COMPATIBILITY

/**
 * Global GPU Configuration Instance
 *
 * Centralized GPU configuration management using modern C++ practices.
 * Encapsulates hardware detection, parameter optimization, and constant
 * memory management in a single, well-defined interface.
 */
static cudamcml::GPUConfig g_gpu_config;

/**
 * Legacy Global Variables - Backward Compatibility
 *
 * These variables maintain compatibility with existing code that expects
 * global GPU parameters. They are automatically synchronized with the
 * modern GPUConfig class during initialization.
 *
 * NOTE: Future code should use the GPUConfig class interface directly
 * rather than these global variables for better encapsulation.
 */
int g_num_blocks = cudamcml::GPUConfig::DEFAULT_FALLBACK_BLOCKS;
int g_threads_per_block = DEFAULT_THREADS_PER_BLOCK;
int g_total_threads = cudamcml::GPUConfig::DEFAULT_FALLBACK_THREADS;

/**
 * Modern GPU Parameter Initialization
 *
 * Initializes GPU configuration based on detected hardware capabilities
 * and synchronizes legacy global variables for backward compatibility.
 * This function should be called once at program startup.
 *
 * INITIALIZATION SEQUENCE:
 * 1. Detect CUDA device capabilities
 * 2. Calculate optimal thread configuration
 * 3. Copy parameters to GPU constant memory
 * 4. Update legacy global variables for compatibility
 * 5. Display configuration report
 */
void initialize_gpu_params() {
	g_gpu_config.initialize();

	// Synchronize legacy globals with modern configuration
	g_num_blocks = g_gpu_config.num_blocks;
	g_threads_per_block = g_gpu_config.threads_per_block;
	g_total_threads = g_gpu_config.total_threads;
}

namespace cudamcml
{

////////////////////////////////////////////////////////////////////////////////
// RANDOM NUMBER GENERATOR MEMORY MANAGEMENT

/**
 * RAII Random Number Generator Memory Manager
 *
 * Manages memory allocation and initialization for multiply-with-carry (MWC)
 * random number generators used in Monte Carlo photon transport. Ensures
 * proper cleanup and exception safety.
 *
 * DESIGN PRINCIPLES:
 * - RAII: Automatic memory management with constructor/destructor
 * - Exception safety: Proper error handling throughout initialization
 * - Performance: Direct memory access for GPU kernel operations
 * - Statistical independence: Each thread gets unique RNG state
 *
 * USAGE:
 *   RNGMemory rng(thread_count);
 *   rng.initialize(seed, "safeprimes_base32.txt");
 *   // Use rng.x() and rng.a() in CUDA kernels
 *   // Automatic cleanup when rng goes out of scope
 */
class RNGMemory
{
public:
	/**
	 * Constructor - Allocate memory for RNG state arrays
	 *
	 * @param thread_count Number of GPU threads requiring RNG state
	 * @throws std::invalid_argument if thread_count <= 0
	 * @throws std::bad_alloc if memory allocation fails
	 */
	explicit RNGMemory(int thread_count) : thread_count_(thread_count) {
		if (thread_count <= 0) {
			throw std::invalid_argument("Thread count must be positive");
		}

		// Allocate memory for RNG state arrays
		x_data = std::make_unique<unsigned long long[]>(thread_count);
		a_data = std::make_unique<unsigned int[]>(thread_count);

		if (!x_data || !a_data) {
			throw std::bad_alloc();
		}
	}

	/**
	 * Initialize RNG state with safe prime multipliers
	 *
	 * Initializes each thread's RNG with unique state values and
	 * safe prime multipliers for good statistical properties.
	 *
	 * @param seed Initial seed value for RNG initialization
	 * @param prime_file Path to safe primes file (or nullptr for embedded primes)
	 * @throws std::runtime_error if RNG initialization fails
	 */
	void initialize(unsigned long long seed, const char* prime_file) {
		const int result = init_rng(x_data.get(), a_data.get(), thread_count_, prime_file, seed);
		if (result != 0) {
			throw std::runtime_error("Failed to initialize RNG with safe primes");
		}
	}

	// Accessor methods for use in CUDA kernel launches
	unsigned long long* x() { return x_data.get(); }   // RNG state array
	unsigned int* a() { return a_data.get(); }         // RNG multiplier array
	int thread_count() const { return thread_count_; } // Number of threads

private:
	int thread_count_;                                 // Number of GPU threads
	std::unique_ptr<unsigned long long[]> x_data;      // RNG state values
	std::unique_ptr<unsigned int[]> a_data;            // RNG multipliers (safe primes)
};

////////////////////////////////////////////////////////////////////////////////
// PERFORMANCE METRICS AND ANALYSIS

/**
 * Performance Metrics Collection and Reporting
 *
 * Tracks simulation performance metrics for throughput analysis and
 * optimization guidance. Provides detailed reporting of GPU utilization
 * and computational efficiency.
 *
 * TRACKED METRICS:
 * - Total simulation time (wall clock)
 * - Total photons processed across all threads
 * - Number of kernel launch iterations
 * - Throughput in photons per second
 * - GPU hardware utilization statistics
 */
struct PerformanceMetrics {
	double simulation_time;           // Total simulation time [seconds]
	unsigned long long total_photons; // Total photons processed
	unsigned int kernel_iterations;   // Number of kernel launches

	/**
	 * Calculate photons per second throughput
	 * @return Throughput in photons per second
	 */
	double photons_per_second() const { return simulation_time > 0.0 ? total_photons / simulation_time : 0.0; }

	/**
	 * Calculate throughput in millions of photons per second
	 * @return Throughput in millions of photons per second (convenient units)
	 */
	double million_photons_per_second() const { return photons_per_second() / 1e6; }

	/**
	 * Print comprehensive performance report
	 * @param gpu_config GPU configuration for context in reporting
	 */
	void print(const GPUConfig& gpu_config) const {
		std::cout << "=== CUDAMCML Performance Report ===\n"
				  << "Total photons simulated: " << total_photons << "\n"
				  << "Simulation time: " << std::fixed << std::setprecision(2) << simulation_time << " sec\n"
				  << "Performance: " << std::fixed << std::setprecision(1) << million_photons_per_second()
				  << " million photons/sec\n"
				  << "GPU utilization: " << gpu_config.num_blocks << " blocks * " << gpu_config.threads_per_block
				  << " threads = " << gpu_config.total_threads << " total threads\n"
				  << "Kernel iterations: " << kernel_iterations << "\n"
				  << "===================================\n\n";
	}
};

} // namespace cudamcml

////////////////////////////////////////////////////////////////////////////////
// MONTE CARLO SIMULATION ORCHESTRATION

namespace cudamcml
{

/**
 * Monte Carlo Simulation Runner
 *
 * Orchestrates the complete multi-layered Monte Carlo photon transport
 * simulation workflow. Manages GPU memory, kernel execution, and result
 * collection for complex tissue geometries.
 *
 * SIMULATION WORKFLOW:
 * 1. Memory initialization: Allocate and initialize GPU memory structures
 * 2. Photon launching: Initialize photon states for all GPU threads
 * 3. Transport simulation: Iterative kernel execution until completion
 * 4. Result collection: Gather detection results and statistics
 * 5. Memory cleanup: Proper deallocation and resource management
 *
 * KEY FEATURES:
 * - Exception-safe memory management using RAII principles
 * - Performance monitoring with detailed timing and throughput metrics
 * - Iterative kernel execution for memory-constrained scenarios
 * - Comprehensive progress reporting and status monitoring
 * - Automatic GPU resource cleanup on completion or error
 */
class SimulationRunner
{
public:
	/**
	 * Constructor - Initialize with GPU configuration
	 * @param gpu_config GPU configuration for kernel launches and memory allocation
	 */
	explicit SimulationRunner(const GPUConfig& gpu_config) : gpu_config_(gpu_config) {}

	/**
	 * Execute Complete Multi-Layered Monte Carlo Simulation
	 *
	 * Runs comprehensive photon transport simulation with the specified
	 * tissue geometry and optical properties. Handles all aspects from
	 * memory management to result collection.
	 *
	 * @param simulation Complete simulation configuration and parameters
	 * @param x Pre-initialized RNG state arrays for all threads
	 * @param a Pre-initialized RNG multiplier arrays for all threads
	 * @return Performance metrics including timing and throughput data
	 */
	PerformanceMetrics run_simulation(SimulationStruct* simulation, unsigned long long* x, unsigned int* a) {
		MemStruct device_mem, host_mem;

		// Phase 1: Initialize GPU and host memory structures
		initialize_memory(simulation, x, a, host_mem, device_mem);

		// Phase 2: Launch initial photon states on all GPU threads
		const clock_t start_time = clock();
		launch_photons(device_mem);

		std::cout << "Absorption detection: " << (simulation->ignoreAdetection ? "disabled" : "enabled") << "\n\n";

		// Phase 3: Execute iterative photon transport simulation
		const unsigned int iterations = run_main_simulation_loop(simulation, device_mem, host_mem);

		std::cout << "Simulation completed successfully!\n";

		// Phase 4: Finalize results and cleanup resources
		finalize_simulation(simulation, host_mem, device_mem);

		// Phase 5: Calculate and return performance metrics
		const clock_t end_time = clock();
		return create_performance_metrics(simulation, start_time, end_time, iterations);
	}

private:
	const GPUConfig& gpu_config_; // Reference to GPU configuration

	/**
	 * Initialize GPU and Host Memory Structures
	 *
	 * Allocates and initializes all memory structures required for the
	 * Monte Carlo simulation, including photon states, RNG arrays, and
	 * detection matrices.
	 *
	 * @param simulation Simulation configuration with geometry and parameters
	 * @param x Pre-initialized RNG state arrays
	 * @param a Pre-initialized RNG multiplier arrays
	 * @param host_mem Host memory structure to initialize
	 * @param device_mem Device memory structure to initialize
	 */
	void initialize_memory(SimulationStruct* simulation, unsigned long long* x, unsigned int* a, MemStruct& host_mem,
						   MemStruct& device_mem) {
		// Connect pre-initialized RNG arrays to memory structure
		host_mem.x = x;
		host_mem.a = a;

		// Initialize all GPU and host memory allocations
		init_mem_structs(&host_mem, &device_mem, simulation);

		// Copy simulation parameters to GPU constant memory
		init_dc_mem(simulation);
	}

	/**
	 * Launch Initial Photon States on All GPU Threads
	 *
	 * Initializes photon position, direction, and weight for all GPU threads
	 * according to the specified source configuration. Each thread receives
	 * a unique photon to begin transport simulation.
	 *
	 * @param device_mem Device memory containing photon state arrays
	 */
	void launch_photons(MemStruct& device_mem) {
		const dim3 dim_block(gpu_config_.threads_per_block);
		const dim3 dim_grid(gpu_config_.num_blocks);

		// Launch photon initialization kernel
		launch_photon_global<<<dim_grid, dim_block>>>(device_mem);
		CUDA_SAFE_CALL(hipDeviceSynchronize());
	}

	/**
	 * Execute Main Iterative Monte Carlo Transport Loop
	 *
	 * Runs iterative photon transport kernels until all photons have been
	 * terminated through absorption, transmission, or Russian roulette.
	 * Provides progress monitoring and adapts to memory constraints.
	 *
	 * ALGORITHM:
	 * 1. Launch transport kernel (with or without absorption detection)
	 * 2. Check thread activity status across all GPU threads
	 * 3. Count terminated photons for progress reporting
	 * 4. Continue until no active threads remain
	 *
	 * @param simulation Simulation configuration for detection mode selection
	 * @param device_mem Device memory containing simulation state
	 * @param host_mem Host memory for thread activity monitoring
	 * @return Number of kernel iterations required for completion
	 */
	unsigned int run_main_simulation_loop(SimulationStruct* simulation, MemStruct& device_mem, MemStruct& host_mem) {
		const dim3 dim_block(gpu_config_.threads_per_block);
		const dim3 dim_grid(gpu_config_.num_blocks);

		unsigned int threads_active_total = 1;
		unsigned int iteration = 0;

		// Continue simulation until all photons are terminated
		while (threads_active_total > 0) {
			++iteration;

			// Launch appropriate transport kernel based on detection configuration
			if (simulation->ignoreAdetection == 1) {
				// Optimized kernel without absorption detection
				mc_d<1><<<dim_grid, dim_block>>>(device_mem);
			}
			else {
				// Full kernel with absorption detection
				mc_d<0><<<dim_grid, dim_block>>>(device_mem);
			}
			CUDA_SAFE_CALL(hipDeviceSynchronize());

			// Monitor progress and thread activity
			threads_active_total = update_thread_activity(device_mem, host_mem);
			const auto terminated_photons = get_terminated_photon_count(device_mem, host_mem);

			std::cout << "Iteration " << iteration << ": Photons terminated = " << terminated_photons
					  << ", Threads active = " << threads_active_total << "\n";
		}

		return iteration;
	}

	/**
	 * Update Thread Activity Status from GPU
	 *
	 * Copies thread activity flags from GPU to host memory and counts
	 * the number of threads still actively transporting photons.
	 *
	 * @param device_mem Device memory containing thread activity flags
	 * @param host_mem Host memory for thread activity monitoring
	 * @return Total number of active threads
	 */
	unsigned int update_thread_activity(MemStruct& device_mem, MemStruct& host_mem) {
		CUDA_SAFE_CALL(hipMemcpy(host_mem.thread_active, device_mem.thread_active,
								  gpu_config_.total_threads * sizeof(unsigned int), hipMemcpyDeviceToHost));

		// Count total active threads across all GPU threads
		unsigned int total_active = 0;
		for (int i = 0; i < gpu_config_.total_threads; ++i) {
			total_active += host_mem.thread_active[i];
		}
		return total_active;
	}

	/**
	 * Get Total Terminated Photon Count from GPU
	 *
	 * Retrieves the global counter of terminated photons for progress
	 * reporting and simulation monitoring.
	 *
	 * @param device_mem Device memory containing terminated photon counter
	 * @param host_mem Host memory for counter retrieval
	 * @return Current number of terminated photons
	 */
	unsigned int get_terminated_photon_count(MemStruct& device_mem, MemStruct& host_mem) {
		CUDA_SAFE_CALL(hipMemcpy(host_mem.num_terminated_photons, device_mem.num_terminated_photons,
								  sizeof(unsigned int), hipMemcpyDeviceToHost));
		return *host_mem.num_terminated_photons;
	}

	/**
	 * Finalize Simulation and Export Results
	 *
	 * Copies final detection results from GPU to host, exports results
	 * to output files, and performs cleanup of GPU memory allocations.
	 *
	 * @param simulation Simulation configuration for output file paths
	 * @param host_mem Host memory structures for result storage
	 * @param device_mem Device memory structures to be cleaned up
	 */
	void finalize_simulation(SimulationStruct* simulation, MemStruct& host_mem, MemStruct& device_mem) {
		// Copy final detection results from GPU to host
		copy_device_to_host_mem(&host_mem, &device_mem, simulation);

		// Export results to standard MCML format files
		write_simulation_results(&host_mem, simulation, 0); // Timing handled externally

		// Clean up GPU memory allocations
		free_mem_structs(&host_mem, &device_mem);
	}

	/**
	 * Create Performance Metrics Report
	 *
	 * Calculates comprehensive performance metrics from simulation timing
	 * and configuration data for throughput analysis.
	 *
	 * @param simulation Simulation configuration with photon count
	 * @param start_time Simulation start timestamp
	 * @param end_time Simulation end timestamp
	 * @param iterations Number of kernel iterations executed
	 * @return Populated performance metrics structure
	 */
	PerformanceMetrics create_performance_metrics(SimulationStruct* simulation, clock_t start_time, clock_t end_time,
												  unsigned int iterations) {
		PerformanceMetrics metrics;
		metrics.simulation_time = static_cast<double>(end_time - start_time) / CLOCKS_PER_SEC;
		metrics.total_photons = simulation->number_of_photons;
		metrics.kernel_iterations = iterations;
		return metrics;
	}
};

} // namespace cudamcml

////////////////////////////////////////////////////////////////////////////////
// LEGACY COMPATIBILITY AND MAIN APPLICATION LOGIC

/**
 * Legacy Wrapper Function - Backward Compatibility
 *
 * Provides compatibility interface for existing code that expects the
 * traditional DoOneSimulation function signature. Internally uses the
 * modern SimulationRunner class for improved error handling and performance.
 *
 * @param simulation Complete simulation configuration
 * @param x Pre-initialized RNG state arrays
 * @param a Pre-initialized RNG multiplier arrays
 */
void do_one_simulation(SimulationStruct* simulation, unsigned long long* x, unsigned int* a) {
	cudamcml::SimulationRunner runner(g_gpu_config);
	const auto metrics = runner.run_simulation(simulation, x, a);
	metrics.print(g_gpu_config);
}

////////////////////////////////////////////////////////////////////////////////
// MAIN ENTRY POINT

/**
 * Main Application Entry Point
 *
 * Provides comprehensive command-line interface for CUDA Monte Carlo
 * multi-layered photon transport simulations. Handles argument parsing,
 * GPU initialization, RNG setup, and simulation execution.
 *
 * EXECUTION WORKFLOW:
 * 1. Parse command line arguments and validate input file
 * 2. Initialize CUDA device and configure optimal GPU parameters
 * 3. Parse simulation input file with tissue geometry and optical properties
 * 4. Initialize random number generators for all GPU threads
 * 5. Execute Monte Carlo photon transport simulation
 * 6. Generate performance report and export results
 * 7. Clean up resources and return appropriate exit codes
 *
 * COMMAND LINE USAGE:
 *   cudamcml <input_file.mci> [options]
 *
 * INPUT FILE FORMAT:
 *   Standard MCML format with multi-layer tissue specifications
 *
 * @param argc Command line argument count
 * @param argv Command line argument array
 * @return EXIT_SUCCESS on successful completion, EXIT_FAILURE on error
 */
int main(int argc, char* argv[]) {
	try {
		std::cout << "=== CUDAMCML - GPU Monte Carlo Multi-Layer Photon Transport ===\n\n";

		// Validate command line arguments
		if (argc < 2) {
			std::cerr << "Usage: " << argv[0] << " <input_file.mci> [options]\n"
					  << "\nRequired:\n"
					  << "  input_file.mci    MCML input file with tissue layer specifications\n"
					  << "\nExample:\n"
					  << "  " << argv[0] << " sample.mci\n\n";
			return EXIT_FAILURE;
		}

		// Initialize CUDA device and configure GPU parameters
		std::cout << "Initializing CUDA device and configuring GPU parameters...\n";
		initialize_gpu_params();

		// Parse simulation input file
		std::cout << "Loading simulation configuration from: " << argv[1] << "\n";
		const char* filename = argv[1];
		auto seed = static_cast<unsigned long long>(time(nullptr));
		int ignoreAdetection = 0;

		// Parse additional command line arguments
		if (interpret_arg(argc, argv, &seed, &ignoreAdetection) != 0) {
			std::cerr << "Error: Invalid command line arguments\n";
			return EXIT_FAILURE;
		}

		// Read simulation data from input file
		SimulationStruct* simulations = nullptr;
		const int n_simulations = read_simulation_data(const_cast<char*>(filename), &simulations, ignoreAdetection);

		if (n_simulations == 0 || !simulations) {
			std::cerr << "Error: Failed to read simulation data from " << filename << "\n";
			return EXIT_FAILURE;
		}

		// Display simulation configuration summary
		std::cout << "\nLoaded " << n_simulations << " simulation(s)\n";
		for (int i = 0; i < n_simulations; ++i) {
			std::cout << "Simulation " << (i + 1) << ":\n"
					  << "  Number of photons: " << simulations[i].number_of_photons << "\n"
					  << "  Number of layers: " << simulations[i].n_layers << "\n"
					  << "  Output file: " << simulations[i].outp_filename << "\n"
					  << "  Absorption detection: " << (simulations[i].ignoreAdetection ? "disabled" : "enabled")
					  << "\n";
		}

		// Initialize random number generators for all GPU threads
		std::cout << "\nInitializing random number generators...\n";
		cudamcml::RNGMemory rng(g_total_threads);
		rng.initialize(seed, nullptr); // Use embedded safe primes

		// Execute all Monte Carlo simulations with performance monitoring
		std::cout << "Starting Monte Carlo photon transport simulation(s)...\n\n";

		for (int i = 0; i < n_simulations; ++i) {
			std::cout << "=== Running Simulation " << (i + 1) << " of " << n_simulations << " ===\n";

			cudamcml::SimulationRunner runner(g_gpu_config);
			const auto metrics = runner.run_simulation(&simulations[i], rng.x(), rng.a());

			// Display performance report for this simulation
			std::cout << "\n";
			metrics.print(g_gpu_config);
			std::cout << "\n";
		}

		// Cleanup simulation configuration
		free_simulation_struct(simulations, n_simulations);

		std::cout << "CUDAMCML simulation completed successfully!\n";
		return EXIT_SUCCESS;
	}
	catch (...) {
		std::cerr << "Fatal error occurred during simulation.\n";
		return EXIT_FAILURE;
	}
}
