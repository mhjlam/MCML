#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// CUDAMC - GPU Monte Carlo for Time-Resolved Photon Transport
//
// GPU Device Functions for CUDAMC Photon Transport
//
// This file contains CUDA device functions that run on the GPU to simulate
// photon transport through homogeneous turbid media with time resolution.
// Functions handle photon initialization, scattering, boundary interactions,
// and detection for time-domain spectroscopy applications.
//
// Authors: Erik Alerstam, Tomas Svensson, Stefan Andersson-Engels
// Modernized: 2025
////////////////////////////////////////////////////////////////////////////////

#include "cudamc.h"

// Define PI constant for scattering calculations
#define PI 3.14159265f

////////////////////////////////////////////////////////////////////////////////
// MAIN MONTE CARLO SIMULATION KERNEL

/**
 * Main GPU kernel for time-resolved Monte Carlo photon transport
 *
 * Each GPU thread simulates multiple photon trajectories through homogeneous
 * tissue with time-resolved detection. Photons undergo scattering, absorption,
 * and boundary interactions while tracking time-of-flight for spectroscopy
 * applications.
 *
 * KERNEL EXECUTION MODEL:
 * -----------------------
 * - One thread per GPU core (typically 512-1024 threads per block)
 * - Each thread simulates ~500,000 photon trajectories
 * - Coalesced memory access for optimal GPU memory bandwidth
 * - Atomic operations for thread-safe histogram accumulation
 *
 * @param rng_states      Per-thread RNG state array
 * @param rng_constant    Per-thread RNG constant array
 * @param rng_multipliers Per-thread RNG multiplier array
 * @param num_device      Device array for storing photon counts per thread
 * @param time_histogram  Time-resolved detection histogram (shared)
 */
__global__ void mc(uint32_t* rng_states_32, uint32_t* rng_constant_32, uint32_t* rng_multipliers,
				   uint32_t* num_device, uint32_t* time_histogram) {
	// Thread identification and memory indexing
	const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

	// Local photon state variables
	float3 photon_position;     // Current photon position [cm]
	float3 photon_direction;    // Current photon direction (unit vector)
	float time_of_flight;       // Total photon time-of-flight [ps]

	// Convert 32-bit RNG arrays to 64-bit for internal use
	uint64_t rng_state = ((uint64_t)rng_constant_32[thread_id] << 32) | rng_states_32[thread_id];
	uint32_t rng_multiplier = rng_multipliers[thread_id];

	// Tissue optical properties (loaded from constant memory)
	float mus_max = 100.0f;    // Maximum scattering coefficient [cm⁻¹]
	float anisotropy_g = 0.9f; // Henyey-Greenberg anisotropy parameter
	float light_speed = 0.0299792458f; // Speed of light in tissue [cm/ps]
	float critical_cos = 0.292f;       // Critical angle cosine for TIR
	float refractive_ratio = 1.4f;     // Refractive index ratio (tissue/air)

	// Detection and simulation counters
	uint32_t num_detected_photons = 0;

	// Launch initial photon state
	launch_photon(&photon_position, &photon_direction, &time_of_flight);

	// Main photon transport loop (500,000 photons per thread)
	for (uint32_t photon_index = 0; photon_index < 500000; photon_index++) {
		// Sample step size from exponential distribution
		float step_size = __fdividef(-__logf(rand_mwc_oc(&rng_state, &rng_multiplier)), mus_max);

		// Move photon and check for boundary crossing
		photon_position.x += photon_direction.x * step_size;
		photon_position.y += photon_direction.y * step_size;
		photon_position.z += photon_direction.z * step_size;

		// Handle boundary interaction if photon exits tissue (z < 0)
		uint32_t boundary_flag = 0;
		if (photon_position.z < 0.0f) {
			boundary_flag = reflect(&photon_direction, &photon_position, &time_of_flight,
									&light_speed, &critical_cos, &refractive_ratio,
									&rng_state, &rng_multiplier, time_histogram);
		}

		// Update time-of-flight
		time_of_flight += __fdividef(step_size, light_speed);

		// Sample new scattering direction
		spin(&photon_direction, &anisotropy_g, &rng_state, &rng_multiplier);

		// Check termination conditions
		if (time_of_flight >= 2000.0f || boundary_flag >= 1) {
			if (boundary_flag == 1) {
				num_detected_photons++;
			}
			// Reset photon for next trajectory
			launch_photon(&photon_position, &photon_direction, &time_of_flight);
		}
	}

	// Store final detection count and convert RNG state back to 32-bit format
	num_device[thread_id] = num_detected_photons;
	rng_states_32[thread_id] = (uint32_t)(rng_state & 0xFFFFFFFFULL);
}

////////////////////////////////////////////////////////////////////////////////
// RANDOM NUMBER GENERATION (GPU DEVICE FUNCTIONS)

/**
 * Multiply-With-Carry RNG: Closed-Open interval [0, 1) - GPU version
 *
 * @param rng_state      RNG state pointer (modified)
 * @param rng_multiplier RNG multiplier pointer
 * @return               Random float in [0, 1)
 */
__device__ float rand_mwc_co(uint64_t* rng_state, uint32_t* rng_multiplier) {
	*rng_state = (*rng_state & 0xffffffffull) * (*rng_multiplier) + (*rng_state >> 32);
	return ((float)((uint32_t)(*rng_state & 0xffffffffull)) / (UINT_MAX));
}

/**
 * Multiply-With-Carry RNG: Open-Closed interval (0, 1] - GPU version
 *
 * @param rng_state      RNG state pointer (modified)
 * @param rng_multiplier RNG multiplier pointer
 * @return               Random float in (0, 1]
 */
__device__ float rand_mwc_oc(uint64_t* rng_state, uint32_t* rng_multiplier) {
	*rng_state = (*rng_state & 0xffffffffull) * (*rng_multiplier) + (*rng_state >> 32);
	return (1.0f - (float)((uint32_t)(*rng_state & 0xffffffffull)) / (UINT_MAX));
}

////////////////////////////////////////////////////////////////////////////////
// PHOTON INITIALIZATION (GPU DEVICE FUNCTIONS)

/**
 * Launch new photon at origin - GPU version
 *
 * @param photon_position  Photon position (modified)
 * @param photon_direction Photon direction (modified)
 * @param time_of_flight   Time of flight (modified)
 */
__device__ void launch_photon(float3* photon_position, float3* photon_direction, float* time_of_flight) {
	// Initialize at tissue surface
	photon_position->x = 0.0f;
	photon_position->y = 0.0f;
	photon_position->z = 0.0f;

	// Direction: straight into tissue (+z)
	photon_direction->x = 0.0f;
	photon_direction->y = 0.0f;
	photon_direction->z = 1.0f;

	// Initialize time
	*time_of_flight = 0.0f;
}

////////////////////////////////////////////////////////////////////////////////
// PHOTON SCATTERING (GPU DEVICE FUNCTIONS)

/**
 * Sample scattering direction using Henyey-Greenberg phase function - GPU version
 *
 * @param photon_direction Current direction (modified)
 * @param anisotropy_g     Anisotropy parameter pointer
 * @param rng_state        RNG state pointer
 * @param rng_multiplier   RNG multiplier pointer
 */
__device__ void spin(float3* photon_direction, float* anisotropy_g, uint64_t* rng_state, uint32_t* rng_multiplier) {
	float cost, sint; // cos(θ) and sin(θ) for polar angle
	float cosp, sinp; // cos(φ) and sin(φ) for azimuthal angle
	float temp, tempdir = photon_direction->x;

	// Sample polar angle from Henyey-Greenberg distribution
	if ((*anisotropy_g) == 0.0f) {
		// Isotropic scattering: uniform cos(θ) in [-1, 1]
		cost = 2.0f * rand_mwc_co(rng_state, rng_multiplier) - 1.0f;
	}
	else {
		// Anisotropic scattering: Henyey-Greenberg phase function
		temp = __fdividef((1.0f - (*anisotropy_g) * (*anisotropy_g)), 
						  (1.0f - (*anisotropy_g) + 2.0f * (*anisotropy_g) * rand_mwc_co(rng_state, rng_multiplier)));
		cost = __fdividef((1.0f + (*anisotropy_g) * (*anisotropy_g) - temp * temp), (2.0f * (*anisotropy_g)));
	}
	sint = sqrtf(1.0f - cost * cost);

	// Sample azimuthal angle uniformly in [0, 2π)
	__sincosf(2.0f * PI * rand_mwc_co(rng_state, rng_multiplier), &sinp, &cosp); // Fast GPU sincos

	// Transform to lab coordinate system
	temp = sqrtf(1.0f - photon_direction->z * photon_direction->z);

	if (temp == 0.0f) {
		// Normal incidence case
		photon_direction->x = sint * cosp;
		photon_direction->y = sint * sinp;
		photon_direction->z = copysignf(cost, photon_direction->z * cost);
	}
	else {
		// General incidence: apply rotation matrix
		photon_direction->x = __fdividef(sint * (photon_direction->x * photon_direction->z * cosp - photon_direction->y * sinp), temp) + photon_direction->x * cost;
		photon_direction->y = __fdividef(sint * (photon_direction->y * photon_direction->z * cosp + tempdir * sinp), temp) + photon_direction->y * cost;
		photon_direction->z = -sint * cosp * temp + photon_direction->z * cost;
	}

	// Renormalize direction vector (GPU-optimized)
	temp = rsqrtf(photon_direction->x * photon_direction->x + photon_direction->y * photon_direction->y + photon_direction->z * photon_direction->z); // Fast inverse sqrt
	photon_direction->x *= temp;
	photon_direction->y *= temp;
	photon_direction->z *= temp;
}

////////////////////////////////////////////////////////////////////////////////
// BOUNDARY REFLECTION (GPU DEVICE FUNCTIONS)

/**
 * Handle boundary interaction at tissue-air interface - GPU version
 *
 * @param photon_direction    Current direction (modified if reflected)
 * @param photon_position     Current position (modified)
 * @param time_of_flight      Time of flight (modified)
 * @param light_speed         Speed of light in medium
 * @param critical_cos        Critical angle cosine
 * @param refractive_ratio    Refractive index ratio
 * @param rng_state           RNG state pointer
 * @param rng_multiplier      RNG multiplier pointer
 * @param time_histogram      Time histogram array
 * @return                    0=reflected, 1=detected, 2=transmitted
 */
__device__ uint32_t reflect(float3* photon_direction, float3* photon_position, float* time_of_flight, float* light_speed, 
							float* critical_cos, float* refractive_ratio, uint64_t* rng_state,
							uint32_t* rng_multiplier, uint32_t* time_histogram) {
	// Detection parameters
	float fibre_separation = 1.0f; // Source-detector separation [cm]
	float fibre_diameter = 0.05f;  // Detector diameter [cm]

	// Calculate Fresnel reflectance
	float r;
	if (-photon_direction->z <= *critical_cos) {
		// Total internal reflection
		r = 1.0f;
	}
	else {
		if (-photon_direction->z == 1.0f) {
			// Normal incidence: simplified Fresnel
			r = __fdividef((1.0f - *refractive_ratio), (1.0f + *refractive_ratio));
			r *= r;
		}
		else {
			// Oblique incidence: full Fresnel calculation
			float sin_angle_i = sqrtf(1.0f - photon_direction->z * photon_direction->z);
			float sin_angle_t = *refractive_ratio * sin_angle_i;
			float cos_angle_t = sqrtf(1.0f - sin_angle_t * sin_angle_t);

			float cos_sum_angle = (-photon_direction->z * cos_angle_t) - sin_angle_i * sin_angle_t;
			float cos_dif_angle = (-photon_direction->z * cos_angle_t) + sin_angle_i * sin_angle_t;
			float sin_sum_angle = sin_angle_i * cos_angle_t + (-photon_direction->z * sin_angle_t);
			float sin_dif_angle = sin_angle_i * cos_angle_t - (-photon_direction->z * sin_angle_t);

			// Average of s and p polarization reflectance
			r = 0.5f * sin_dif_angle * sin_dif_angle
				* __fdividef((cos_dif_angle * cos_dif_angle + cos_sum_angle * cos_sum_angle),
							 (sin_sum_angle * sin_sum_angle * cos_dif_angle * cos_dif_angle));
		}
	}

	// Monte Carlo reflection/transmission decision
	if (r < 1.0f) {
		if (rand_mwc_co(rng_state, rng_multiplier) <= r) {
			// Photon reflects
			r = 1.0f;
		}
		else {
			// Photon transmits: calculate exit position and check detection
			r = __fdividef(-photon_position->z, photon_direction->z); // Time to reach boundary
			photon_position->x += photon_direction->x * r;
			photon_position->y += photon_direction->y * r;
			*time_of_flight += __fdividef(r, *light_speed);         // Update total time of flight

			// Calculate radial distance from source at exit
			r = sqrtf(photon_position->x * photon_position->x + photon_position->y * photon_position->y);

			// Check for fiber detection
			if (fabsf(r - fibre_separation) <= fibre_diameter) {
				// Photon detected: add to time-resolved histogram
				uint32_t bin = __float2uint_rz(__fdividef((*time_of_flight), DT));
				if (bin < TEMP_SIZE) {               // Bounds check
					atomicAdd(time_histogram + bin, 1); // Thread-safe increment
				}
				return 1;                            // Detected
			}
			else {
				return 2;                            // Transmitted but not detected
			}
		}
	}

	// Handle reflection back into medium
	if (r == 1.0f) {
		photon_position->z *= -1; // Mirror z-position across boundary
		photon_direction->z *= -1; // Mirror z-direction component
	}

	return 0;         // Reflected
}
