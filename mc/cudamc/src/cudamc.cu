#include "hip/hip_runtime.h"
/*==============================================================================
 * CUDAMC - GPU-Accelerated Monte Carlo Photon Transport Simulation
 *
 * PHYSICAL MODEL:
 * ---------------
 * - Semi-infinite turbid medium with specified optical properties
 * - Isotropic point source at medium surface (z=0)
 * - Henyey-Greenberg phase function for anisotropic scattering
 * - Fresnel reflection/transmission at medium boundaries
 * - Time-resolved photon detection with binned histogram output
 * - Statistical independence via multiply-with-carry random number generation
 *
 * COMPUTATIONAL ARCHITECTURE:
 * ---------------------------
 * - GPU Implementation: CUDA kernels with massive parallelization
 * - CPU Reference: "Gold standard" validation with identical algorithms
 * - Modern C++: RAII memory management, exception handling, namespaces
 * - Performance Analysis: Comprehensive throughput and speedup metrics
 * - Static Configuration: Hardware-agnostic thread allocation
 *
 * SIMULATION FEATURES:
 * --------------------
 * - Concurrent photon histories on GPU (65,536 threads typical)
 * - Time-resolved detection with configurable bin resolution
 * - Comprehensive validation against CPU reference implementation
 * - Performance benchmarking with detailed throughput analysis
 * - Error handling with CUDA-specific diagnostics
 *
 * USAGE:
 * ------
 * Compile with nvcc and run directly. The simulation automatically:
 * 1. Detects and configures available CUDA hardware
 * 2. Runs GPU simulation with massive parallelization
 * 3. Validates results against CPU reference implementation
 * 4. Reports comprehensive performance metrics and speedup factors
 */

////////////////////////////////////////////////////////////////////////////////
// SYSTEM INCLUDES AND DEPENDENCIES

// Standard C++ library includes for I/O, memory management, and error handling
#include <cstdio>    // C-style I/O for compatibility
#include <iomanip>   // Stream manipulators for formatted output
#include <iostream>  // Modern C++ I/O streams
#include <memory>    // Smart pointers for RAII
#include <stdexcept> // Standard exception classes

// CUDA runtime and device includes for GPU computing
#include <hip/hip_runtime.h>             // CUDA runtime API
#include <> // CUDA kernel launch parameters
#include <hip/hip_math_constants.h>           // Mathematical constants (M_PI, etc.)

// Project-specific includes
#include "cudamc.h"      // Main header with constants and function declarations
#include "safe_primes.h" // Embedded prime numbers for RNG initialization

// Component modules - included as source for single compilation unit
#include "cudamc_gold_standard.c" // CPU reference implementation
#include "cudamc_transport.cu"    // GPU device functions and kernels

////////////////////////////////////////////////////////////////////////////////
// CUDA MONTE CARLO NAMESPACE

namespace cudamc
{

////////////////////////////////////////////////////////////////////////////////
// GPU CONFIGURATION AND HARDWARE MANAGEMENT

/**
 * GPU Configuration Management Class
 *
 * Encapsulates CUDA hardware detection, configuration, and optimization.
 * Uses static thread allocation to match global array sizes and ensure
 * consistent memory access patterns across different GPU architectures.
 *
 * DESIGN PHILOSOPHY:
 * - Static configuration for predictable memory footprint
 * - Hardware detection for informational purposes and validation
 * - Conservative thread allocation for broad hardware compatibility
 * - Detailed reporting for performance analysis and debugging
 */
class CPUConfig
{
public:
	// Fallback configuration constants for hardware compatibility
	static constexpr int DEFAULT_FALLBACK_BLOCKS = 128;
	static constexpr int DEFAULT_FALLBACK_THREADS = 65536;

	// Active GPU configuration parameters
	int num_blocks;        // Number of CUDA thread blocks
	int threads_per_block; // Threads per block (typically 256)
	int total_threads;     // Total concurrent threads

	/**
	 * Constructor - Initialize with static configuration
	 *
	 * Uses compile-time constants to ensure consistent memory allocation
	 * and avoid runtime configuration complexity across different GPUs.
	 */
	CPUConfig() : num_blocks(NUM_BLOCKS), threads_per_block(NUM_THREADS_PER_BLOCK), total_threads(NUM_THREADS) {}

	/**
	 * Hardware Detection and Configuration Validation
	 *
	 * Detects GPU capabilities and validates our static configuration
	 * against hardware limits. Provides detailed hardware information
	 * for performance analysis and troubleshooting.
	 */
	void initialize() {
		hipDeviceProp_t device_prop;
		int device;

		CUDA_CHECK_ERROR(hipGetDevice(&device));
		CUDA_CHECK_ERROR(hipGetDeviceProperties(&device_prop, device));

		// Extract hardware characteristics for reporting
		const int multiprocessors = device_prop.multiProcessorCount;
		const int max_threads_per_sm = device_prop.maxThreadsPerMultiProcessor;
		const int max_blocks_per_sm = device_prop.maxBlocksPerMultiProcessor;

		// Use static configuration to match global array allocations
		// This ensures consistent behavior across different GPU models
		threads_per_block = NUM_THREADS_PER_BLOCK;
		num_blocks = NUM_BLOCKS;
		total_threads = NUM_THREADS;

		print_configuration(device_prop, multiprocessors, max_threads_per_sm, max_blocks_per_sm);
	}

private:
	/**
	 * Print comprehensive GPU configuration report
	 *
	 * Displays hardware capabilities, configured parameters, and
	 * utilization analysis for performance optimization guidance.
	 */
	void print_configuration(const hipDeviceProp_t& device_prop, int multiprocessors, int max_threads_per_sm,
							 int max_blocks_per_sm) const {
		std::cout << "=== CUDAMC GPU Configuration ===\n"
				  << "GPU: " << device_prop.name << "\n"
				  << "Compute capability: " << device_prop.major << "." << device_prop.minor << "\n"
				  << "Multiprocessors: " << multiprocessors << "\n"
				  << "Max threads per SM: " << max_threads_per_sm << "\n"
				  << "Max blocks per SM: " << max_blocks_per_sm << "\n"
				  << "Configured blocks: " << num_blocks << "\n"
				  << "Threads per block: " << threads_per_block << "\n"
				  << "Total GPU threads: " << total_threads << "\n"
				  << "Configuration: Static (matching array sizes)\n"
				  << "=================================\n\n";
	}
};

////////////////////////////////////////////////////////////////////////////////
// RAII DEVICE MEMORY MANAGEMENT

/**
 * RAII Device Memory Wrapper Template
 *
 * Provides automatic GPU memory management using Resource Acquisition Is
 * Initialization (RAII) principles. Ensures proper cleanup even in the
 * presence of exceptions, preventing GPU memory leaks.
 *
 * FEATURES:
 * - Automatic allocation and deallocation
 * - Exception-safe memory management
 * - Move semantics for efficient transfers
 * - Type-safe memory operations
 * - Convenient host-device data transfer methods
 *
 * USAGE:
 * DeviceMemory<float> gpu_array(1000);           // Allocate 1000 floats on GPU
 * gpu_array.copy_from_host(host_data);           // Transfer from CPU
 * kernel<<<blocks, threads>>>(gpu_array.get());  // Use in kernel
 * gpu_array.copy_to_host(results);               // Transfer results back
 * // Automatic cleanup when gpu_array goes out of scope
 */
template<typename T>
class DeviceMemory
{
public:
	/**
	 * Constructor - Allocate GPU memory for specified number of elements
	 *
	 * @param count Number of elements to allocate (not bytes)
	 * @throws std::invalid_argument if count is zero
	 * @throws std::runtime_error if GPU allocation fails
	 */
	explicit DeviceMemory(size_t count) : count_(count), size_(count * sizeof(T)) {
		if (count == 0) {
			throw std::invalid_argument("Memory count must be positive");
		}

		CUDA_CHECK_ERROR(hipMalloc(reinterpret_cast<void**>(&device_ptr_), size_));
	}

	/**
	 * Destructor - Automatic GPU memory deallocation
	 *
	 * Provides exception-safe cleanup. CUDA errors during deallocation
	 * are not propagated to avoid termination during stack unwinding.
	 */
	~DeviceMemory() {
		if (device_ptr_) {
			hipFree(device_ptr_); // Ignore errors in destructor
		}
	}

	// Non-copyable to prevent accidental double-free errors
	DeviceMemory(const DeviceMemory&) = delete;
	DeviceMemory& operator=(const DeviceMemory&) = delete;

	/**
	 * Move constructor - Transfer ownership efficiently
	 *
	 * Enables returning DeviceMemory objects from functions and
	 * storing them in containers without unnecessary allocations.
	 */
	DeviceMemory(DeviceMemory&& other) noexcept :
		device_ptr_(other.device_ptr_), count_(other.count_), size_(other.size_) {
		other.device_ptr_ = nullptr;
		other.count_ = 0;
		other.size_ = 0;
	}

	/**
	 * Copy data from host to device
	 *
	 * @param host_data Pointer to host memory containing source data
	 * @throws std::runtime_error if CUDA memory copy fails
	 */
	void copy_from_host(const T* host_data) {
		CUDA_CHECK_ERROR(hipMemcpy(device_ptr_, host_data, size_, hipMemcpyHostToDevice));
	}

	/**
	 * Copy data from device to host
	 *
	 * @param host_data Pointer to host memory for storing results
	 * @throws std::runtime_error if CUDA memory copy fails
	 */
	void copy_to_host(T* host_data) const {
		CUDA_CHECK_ERROR(hipMemcpy(host_data, device_ptr_, size_, hipMemcpyDeviceToHost));
	}

	// Accessor methods for use in CUDA kernel launches and API calls
	T* get() const { return device_ptr_; }  // Get raw device pointer
	size_t count() const { return count_; } // Get element count
	size_t size() const { return size_; }   // Get size in bytes

private:
	T* device_ptr_ = nullptr;               // Raw CUDA device pointer
	size_t count_;                          // Number of elements allocated
	size_t size_;                           // Total size in bytes
};

////////////////////////////////////////////////////////////////////////////////
// PERFORMANCE METRICS AND BENCHMARKING

/**
 * Performance Metrics Collection and Analysis
 *
 * Comprehensive performance tracking for both GPU and CPU implementations,
 * enabling detailed analysis of acceleration benefits and computational
 * efficiency. Tracks multiple performance indicators beyond simple timing.
 *
 * METRICS TRACKED:
 * - Execution time (wall clock) for both GPU and CPU
 * - Total computational operations performed
 * - Number of photons that terminated (reached detector)
 * - Histogram contributions (time-resolved detections)
 * - Throughput in billions of operations per second (GOPS)
 * - GPU vs CPU speedup factor with load normalization
 *
 * ANALYSIS FEATURES:
 * - Normalized throughput calculations accounting for different workloads
 * - Comprehensive speedup analysis with proper operation counting
 * - Detailed performance reporting with hardware utilization metrics
 */
struct PerformanceMetrics {
	// Timing measurements (wall clock seconds)
	double gpu_time_seconds;
	double cpu_time_seconds;

	// Operation counting for throughput analysis
	double total_gpu_operations; // Total GPU computational operations
	double total_cpu_operations; // Total CPU computational operations

	// Physics simulation results for validation
	uint64_t gpu_photons_terminated;      // GPU photons reaching detector
	uint64_t cpu_photons_terminated;      // CPU photons reaching detector
	uint64_t gpu_histogram_contributions; // GPU time-resolved detections
	uint64_t cpu_histogram_contributions; // CPU time-resolved detections

	/**
	 * Calculate GPU throughput in billions of operations per second
	 * @return GOPS (billions of operations per second) for GPU execution
	 */
	double gpu_throughput_gops() const {
		return gpu_time_seconds > 0.0 ? total_gpu_operations / gpu_time_seconds / 1e9 : 0.0;
	}

	/**
	 * Calculate CPU throughput in billions of operations per second
	 * @return GOPS (billions of operations per second) for CPU execution
	 */
	double cpu_throughput_gops() const {
		return cpu_time_seconds > 0.0 ? total_cpu_operations / cpu_time_seconds / 1e9 : 0.0;
	}

	/**
	 * Calculate normalized GPU vs CPU speedup factor
	 *
	 * Accounts for different workload sizes by normalizing based on
	 * operations performed rather than just execution time.
	 *
	 * @return Speedup factor (how many times faster GPU is than CPU)
	 */
	double speedup_factor() const {
		return (cpu_time_seconds > 0.0 && gpu_time_seconds > 0.0)
				   ? (total_gpu_operations * cpu_time_seconds) / (total_cpu_operations * gpu_time_seconds)
				   : 0.0;
	}

	/**
	 * Print comprehensive performance analysis report
	 *
	 * Generates detailed performance report including individual GPU/CPU
	 * metrics, comparison analysis, and hardware utilization information.
	 *
	 * @param config GPU configuration for context in reporting
	 */
	void print(const CPUConfig& config) const {
		std::cout << "=== CUDAMC Performance Report ===\n";

		// GPU Performance Analysis
		std::cout << "\nGPU Performance:\n"
				  << "  Simulation time: " << std::fixed << std::setprecision(3) << gpu_time_seconds << " sec\n"
				  << "  Total operations: " << std::scientific << std::setprecision(2) << total_gpu_operations << "\n"
				  << "  Throughput: " << std::fixed << std::setprecision(2) << gpu_throughput_gops()
				  << " billion ops/sec\n"
				  << "  Photons terminated: " << gpu_photons_terminated << "\n"
				  << "  Histogram contributions: " << gpu_histogram_contributions << "\n";

		// CPU Performance Analysis
		std::cout << "\nCPU Performance:\n"
				  << "  Simulation time: " << std::fixed << std::setprecision(3) << cpu_time_seconds << " sec\n"
				  << "  Total operations: " << std::scientific << std::setprecision(2) << total_cpu_operations << "\n"
				  << "  Throughput: " << std::fixed << std::setprecision(2) << cpu_throughput_gops()
				  << " billion ops/sec\n"
				  << "  Photons terminated: " << cpu_photons_terminated << "\n"
				  << "  Histogram contributions: " << cpu_histogram_contributions << "\n";

		// Comparative Performance Analysis
		std::cout << "\nPerformance Comparison:\n"
				  << "  GPU vs CPU speedup: " << std::fixed << std::setprecision(1) << speedup_factor() << "x\n"
				  << "  GPU utilization: " << config.num_blocks << " blocks * " << config.threads_per_block
				  << " threads = " << config.total_threads << " total threads\n"
				  << "==================================\n\n";
	}
};

} // namespace cudamc

////////////////////////////////////////////////////////////////////////////////
// GLOBAL STATE MANAGEMENT AND RNG INITIALIZATION

// Global GPU configuration instance
static cudamc::CPUConfig g_gpu_config;

// Global RNG arrays for all simulation threads
// These arrays are shared between GPU and CPU implementations
uint32_t x_test[NUM_THREADS]; // RNG state values (lower 32 bits)
uint32_t c_test[NUM_THREADS]; // RNG carry values
uint32_t a_test[NUM_THREADS]; // RNG multiplier values (from safe primes)

/**
 * Initialize Random Number Generators for All Threads
 *
 * Sets up independent RNG streams for each simulation thread using
 * multiply-with-carry (MWC) generators with safe prime multipliers.
 * Ensures statistical independence between threads while maintaining
 * good random number quality.
 *
 * ALGORITHM:
 * 1. Load initial multiplier from embedded safe primes data
 * 2. Generate unique initial state for each thread using MWC sequence
 * 3. Assign unique safe prime multiplier to each thread
 * 4. Initialize carry values using normalized random values
 *
 * THREAD SAFETY:
 * Each thread gets independent RNG state, ensuring no correlation
 * between parallel Monte Carlo histories on different threads.
 */
void initialize_rng() {
	uint32_t begin = 0U;
	uint64_t x_init = 1ULL; // Initial seed for state generation
	uint32_t c_init = 0U;   // Initial carry value
	uint32_t for_a;

	// Load first safe prime as base multiplier for state generation
	begin = safeprimes_data[0].a;

	// Generate unique RNG state for each thread
	for (uint32_t i = 0; i < NUM_THREADS; i++) {
		// Generate next x value using MWC with base multiplier
		x_init = x_init * begin + c_init;
		c_init = x_init >> 32;           // Extract carry
		x_init = x_init & 0xffffffffULL; // Keep lower 32 bits
		x_test[i] = static_cast<uint32_t>(x_init);

		// Assign unique safe prime multiplier to this thread
		// Cycle through available safe primes to ensure coverage
		for_a = safeprimes_data[(i + 1) % 50000].a; // Use actual safeprimes count
		a_test[i] = for_a;

		// Generate carry value for this thread
		x_init = x_init * begin + c_init;
		c_init = x_init >> 32;
		x_init = x_init & 0xffffffffULL;
		// Scale carry to appropriate range for this multiplier
		c_test[i] = static_cast<uint32_t>((static_cast<double>(x_init) / UINT_MAX) * for_a);
	}
}

namespace cudamc
{

////////////////////////////////////////////////////////////////////////////////
// MONTE CARLO SIMULATION ORCHESTRATION

/**
 * Monte Carlo Simulation Runner
 *
 * Orchestrates comprehensive Monte Carlo photon transport simulations
 * on both GPU and CPU platforms. Manages memory allocation, kernel
 * execution, result validation, and performance analysis.
 *
 * SIMULATION WORKFLOW:
 * 1. GPU Simulation: Massively parallel photon histories on CUDA
 * 2. CPU Reference: Sequential validation using identical algorithms
 * 3. Results Comparison: Validation of GPU implementation accuracy
 * 4. Performance Analysis: Comprehensive throughput and speedup metrics
 *
 * KEY FEATURES:
 * - RAII memory management for exception safety
 * - Comprehensive error handling with detailed diagnostics
 * - Performance profiling with operation counting
 * - Results validation between GPU and CPU implementations
 * - Detailed progress reporting and logging
 */
class SimulationRunner
{
public:
	/**
	 * Constructor - Initialize with GPU configuration
	 * @param config GPU configuration parameters for kernel launch
	 */
	explicit SimulationRunner(const CPUConfig& config) : gpu_config_(config) {}

	/**
	 * Execute Complete Monte Carlo Simulation Suite
	 *
	 * Runs both GPU and CPU Monte Carlo simulations with identical
	 * parameters and algorithms. Provides comprehensive performance
	 * analysis and validation of results.
	 *
	 * @param x RNG state arrays (lower 32 bits per thread)
	 * @param c RNG carry arrays per thread
	 * @param a RNG multiplier arrays per thread
	 * @return Performance metrics for both GPU and CPU executions
	 */
	PerformanceMetrics run_monte_carlo(uint32_t* x, uint32_t* c, uint32_t* a) {
		PerformanceMetrics metrics {};

		// Execute GPU simulation with massive parallelization
		run_gpu_simulation(x, c, a, metrics);

		// Execute CPU reference implementation for validation
		run_cpu_simulation(x, c, a, metrics);

		return metrics;
	}

private:
	const CPUConfig& gpu_config_; // GPU configuration reference

	/**
	 * Execute GPU Monte Carlo Simulation
	 *
	 * Launches CUDA kernel with thousands of parallel threads, each
	 * simulating independent photon histories. Uses RAII for memory
	 * management and comprehensive error checking.
	 *
	 * EXECUTION PHASES:
	 * 1. Host memory allocation and initialization
	 * 2. Device memory allocation and data transfer
	 * 3. CUDA kernel launch with configured thread geometry
	 * 4. Result retrieval and performance measurement
	 * 5. Automatic cleanup via RAII
	 *
	 * @param x RNG state arrays
	 * @param c RNG carry arrays
	 * @param a RNG multiplier arrays
	 * @param metrics Output metrics structure to populate
	 */
	void run_gpu_simulation(uint32_t* x, uint32_t* c, uint32_t* a, PerformanceMetrics& metrics) {
		std::cout << "Running GPU simulation...\n";
		std::cout << "GPU simulation: " << gpu_config_.total_threads << " threads * " << NUM_STEPS_GPU
				  << " steps = " << std::scientific << (double)gpu_config_.total_threads * NUM_STEPS_GPU
				  << " total operations\n";

		// Allocate host memory for results using smart pointers
		auto num = std::make_unique<uint32_t[]>(gpu_config_.total_threads);
		auto hist = std::make_unique<uint32_t[]>(TEMP_SIZE);

		// Initialize time-resolved histogram bins
		for (uint32_t i = 0; i < TEMP_SIZE; i++) {
			hist[i] = 0;
		}

		// GPU memory management with RAII - automatic cleanup guaranteed
		DeviceMemory<uint32_t> x_device(gpu_config_.total_threads);
		DeviceMemory<uint32_t> c_device(gpu_config_.total_threads);
		DeviceMemory<uint32_t> a_device(gpu_config_.total_threads);
		DeviceMemory<uint32_t> num_device(gpu_config_.total_threads);
		DeviceMemory<uint32_t> hist_device(TEMP_SIZE);

		// Transfer RNG initialization data and histogram to GPU
		x_device.copy_from_host(x);
		c_device.copy_from_host(c);
		a_device.copy_from_host(a);
		hist_device.copy_from_host(hist.get());

		// Begin performance timing for GPU execution
		const clock_t time1 = clock();

		// Configure CUDA kernel launch parameters
		const dim3 dim_block(gpu_config_.threads_per_block); // Threads per block
		const dim3 dim_grid(gpu_config_.num_blocks);         // Number of blocks

		// Ensure GPU is synchronized before kernel launch
		CUDA_CHECK_ERROR(hipDeviceSynchronize());

		std::cout << "Launching kernel with " << gpu_config_.num_blocks << " blocks * " << gpu_config_.threads_per_block
				  << " threads = " << gpu_config_.total_threads << " total threads\n";

		// Launch Monte Carlo kernel - each thread simulates independent photon histories
		// Kernel signature: mc(x_device, c_device, a_device, num_device, hist_device)
		mc<<<dim_grid, dim_block>>>(x_device.get(), c_device.get(), a_device.get(), num_device.get(),
									hist_device.get());

		// Comprehensive error checking for kernel launch and execution
		CUDA_CHECK_ERROR(hipGetLastError());      // Check for launch errors
		CUDA_CHECK_ERROR(hipDeviceSynchronize()); // Wait for completion and check execution errors

		// Retrieve simulation results from GPU to host memory
		num_device.copy_to_host(num.get());   // Per-thread photon termination counts
		hist_device.copy_to_host(hist.get()); // Time-resolved detection histogram

		// End performance timing
		const clock_t time2 = clock();

		// Calculate comprehensive GPU performance metrics
		metrics.gpu_time_seconds = static_cast<double>(time2 - time1) / CLOCKS_PER_SEC;
		metrics.total_gpu_operations = static_cast<double>(gpu_config_.total_threads) * NUM_STEPS_GPU;

		// Count total photons that reached the detector (terminated)
		metrics.gpu_photons_terminated = 0;
		for (uint32_t i = 0; i < gpu_config_.total_threads; i++) {
			metrics.gpu_photons_terminated += num[i];
		}

		// Count total time-resolved detection events across all histogram bins
		metrics.gpu_histogram_contributions = 0;
		for (uint32_t i = 0; i < TEMP_SIZE; i++) {
			metrics.gpu_histogram_contributions += hist[i];
		}

		// Display GPU time-resolved detection histogram
		std::cout << "\nGPU Histogram: ";
		for (uint32_t i = 0; i < TEMP_SIZE; i++) {
			std::cout << hist[i] << " ";
		}
		std::cout << "\n";
	}

	/**
	 * Execute CPU Reference Monte Carlo Simulation
	 *
	 * Runs identical Monte Carlo algorithm on CPU for validation and
	 * performance comparison. Uses the same RNG initialization and
	 * physics parameters to ensure comparable results.
	 *
	 * VALIDATION PURPOSE:
	 * - Verify correctness of GPU implementation
	 * - Provide performance baseline for speedup calculations
	 * - Debug algorithm differences between GPU and CPU
	 *
	 * @param x RNG state arrays (same as used for GPU)
	 * @param c RNG carry arrays (same as used for GPU)
	 * @param a RNG multiplier arrays (same as used for GPU)
	 * @param metrics Output metrics structure to populate
	 */
	void run_cpu_simulation(uint32_t* x, uint32_t* c, uint32_t* a, PerformanceMetrics& metrics) {
		std::cout << "\nRunning CPU simulation (this may take several minutes)...\n";
		std::cout << "CPU simulation: " << NUM_THREADS_CPU << " threads * " << NUM_STEPS_CPU
				  << " steps = " << std::scientific << (double)NUM_THREADS_CPU * NUM_STEPS_CPU << " total operations\n";

		// Allocate host arrays for CPU results
		auto num_h = std::make_unique<uint32_t[]>(NUM_THREADS_CPU);
		auto hist_h = std::make_unique<uint32_t[]>(TEMP_SIZE);

		// Initialize CPU histogram bins
		for (uint32_t i = 0; i < TEMP_SIZE; i++) {
			hist_h[i] = 0;
		}

		// Execute CPU Monte Carlo simulation with performance timing
		const clock_t time1 = clock();
		gs_mc(x, c, a, num_h.get(), hist_h.get()); // Gold standard CPU implementation
		const clock_t time2 = clock();

		// Calculate comprehensive CPU performance metrics
		metrics.cpu_time_seconds = static_cast<double>(time2 - time1) / CLOCKS_PER_SEC;
		metrics.total_cpu_operations = static_cast<double>(NUM_THREADS_CPU) * NUM_STEPS_CPU;

		// Count total photons that reached the detector on CPU
		metrics.cpu_photons_terminated = 0;
		for (uint32_t i = 0; i < NUM_THREADS_CPU; i++) {
			metrics.cpu_photons_terminated += num_h[i];
		}

		// Count total time-resolved detection events across all histogram bins
		metrics.cpu_histogram_contributions = 0;
		for (uint32_t i = 0; i < TEMP_SIZE; i++) {
			metrics.cpu_histogram_contributions += hist_h[i];
		}

		// Display CPU time-resolved detection histogram for comparison
		std::cout << "\nCPU Histogram: ";
		for (uint32_t i = 0; i < TEMP_SIZE; i++) {
			std::cout << hist_h[i] << " ";
		}
		std::cout << "\n";
	}
};

} // namespace cudamc

////////////////////////////////////////////////////////////////////////////////
// MAIN APPLICATION CONTROL AND COORDINATION

/**
 * Main Monte Carlo Simulation Coordinator
 *
 * Orchestrates the complete CUDA Monte Carlo photon transport simulation
 * workflow. Handles CUDA device management, simulation execution, and
 * comprehensive performance reporting.
 *
 * SIMULATION WORKFLOW:
 * 1. CUDA Device Detection and Initialization
 * 2. GPU Hardware Configuration and Optimization
 * 3. Parallel GPU Monte Carlo Simulation
 * 4. Sequential CPU Reference Validation
 * 5. Performance Analysis and Detailed Reporting
 *
 * ERROR HANDLING:
 * - CUDA device availability validation
 * - Comprehensive error reporting with exceptions
 * - Resource cleanup guaranteed via RAII
 *
 * @param x Pre-initialized RNG state arrays for all threads
 * @param c Pre-initialized RNG carry arrays for all threads
 * @param a Pre-initialized RNG multiplier arrays for all threads
 */
void run_monte_carlo_simulation(uint32_t* x, uint32_t* c, uint32_t* a) {
	try {
		// Validate CUDA device availability before proceeding
		int device_count;
		CUDA_CHECK_ERROR(hipGetDeviceCount(&device_count));

		if (device_count == 0) {
			throw std::runtime_error("No CUDA devices found");
		}

		// Initialize GPU hardware configuration and display capabilities
		g_gpu_config.initialize();

		// Execute complete simulation suite (GPU + CPU validation)
		cudamc::SimulationRunner runner(g_gpu_config);
		const auto metrics = runner.run_monte_carlo(x, c, a);

		// Generate comprehensive performance analysis report
		metrics.print(g_gpu_config);

		std::cout << "Monte Carlo simulation completed successfully!\n";
	}
	catch (const std::exception& e) {
		std::cerr << "Error in Monte Carlo simulation: " << e.what() << "\n";
		throw; // Re-throw for higher-level handling
	}
}

////////////////////////////////////////////////////////////////////////////////
// MAIN ENTRY POINT

/**
 * Main Application Entry Point
 *
 * Provides comprehensive exception handling and coordinates the complete
 * CUDAMC simulation workflow from initialization through final reporting.
 *
 * EXECUTION PHASES:
 * 1. Display application banner and initialization
 * 2. Initialize random number generators for all threads
 * 3. Execute complete Monte Carlo simulation suite
 * 4. Handle any errors with detailed diagnostics
 * 5. Return appropriate exit codes
 *
 * @param argc Command line argument count (currently unused)
 * @param argv Command line arguments (currently unused)
 * @return EXIT_SUCCESS on successful completion, EXIT_FAILURE on error
 */
int main(int argc, char* argv[]) {
	try {
		std::cout << "=== CUDAMC Monte Carlo Photon Migration Simulation ===\n\n";

		// Initialize random number generators for all simulation threads
		initialize_rng();

		// Execute comprehensive Monte Carlo simulation and analysis
		run_monte_carlo_simulation(x_test, c_test, a_test);

		return EXIT_SUCCESS;
	}
	catch (const std::exception& e) {
		std::cerr << "Fatal error: " << e.what() << "\n";
		return EXIT_FAILURE;
	}
	catch (...) {
		std::cerr << "Unknown fatal error occurred.\n";
		return EXIT_FAILURE;
	}
}
